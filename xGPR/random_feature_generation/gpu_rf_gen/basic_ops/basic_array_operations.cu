#include "hip/hip_runtime.h"
/*
* Contains all functions needed to run the structured orthogonal features
* (SORF) and randomized Hadamard transform (RHT) operations on an input 3d
* array on GPU. Note that many operations here assume specific dimensions
* of the input array are a power of 2. The Cython wrapper checks this, so do
* not call these routines OUTSIDE of the Cython wrapper -- use the Cython wrapper.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <complex.h>
#include "basic_array_operations.h"
#include "sharedmem.h"

#define DEFAULT_THREADS_PER_BLOCK 256
#define MAX_BASE_LEVEL_TRANSFORM 512



//Uses shared memory to perform a reasonably efficient single kernel
//transform covering strides up to MAX_BASE_LEVEL_TRANSFORM.
template <typename T>
__global__ void baseLevelTransform(T cArray[], int N, int log2N){
    int startPos = blockIdx.x << log2N;

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int i, spacing, pos = threadIdx.x;
    int lo, id1, id2;
    T *src_ptr = cArray + startPos;
    T y;

    for (i = threadIdx.x; i < N; i += blockDim.x)
        s_data[i] = src_ptr[i];

    id1 = (pos << 1);
    id2 = id1 + 1;
    __syncthreads();
    y = s_data[id2];
    s_data[id2] = s_data[id1] - y;
    s_data[id1] += y;


    for (spacing = 2; spacing < N; spacing <<= 1){
        //Equivalent to pos mod spacing if spacing is a power of 2,
        //which here is always true.
        lo = pos & (spacing - 1);
        id1 = ((pos - lo) << 1) + lo;
        id2 = id1 + spacing;
        __syncthreads();
        y = s_data[id2];
        s_data[id2] = s_data[id1] - y;
        s_data[id1] += y;
    }
    for (i = threadIdx.x; i < N; i += blockDim.x){
        src_ptr[i] = s_data[i];
    }
}



//Performs subsequent stages of the transform (strides
//> MAX_BASE_LEVEL_TRANSFORM) using a less efficient global
//memory procedure.
template <typename T>
__global__ void levelNTransform(T cArray[], int arrsize,
                                int spacing)
{
    int pos = blockDim.x * blockIdx.x + threadIdx.x;
    //Equivalent to pos mod spacing if spacing is a power of 2,
    //which here is always true.
    int lo = (pos & (spacing - 1));
    int id = lo + ((pos - lo) << 1);
    
    if (id < arrsize){
        T y, *cPtr = cArray + id;

        y = cPtr[spacing];
        cPtr[spacing] = *cPtr - y;
        *cPtr += y;
    }
}


//Performs an elementwise multiplication of a [c,M,P] array against the
//[N,M,P] input array or a [P] array against the [N,P] input array.
//Note that the last dimensions of these must be the
//same, and this function does not check this -- caller must check. Note that
//we mutiiply by the Hadamard normalization constant here.
template <typename T>
__global__ void diagonalRademMultiply(T cArray[], const int8_t *rademArray,
			int numElementsPerRow, int numElements, T normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rVal, position;
    
    position = tid % numElementsPerRow;

    if (tid < numElements){
        rVal = rademArray[position];
        cArray[tid] = cArray[tid] * rVal * normConstant;
    }
}


//Performs an elementwise multiplication of a row of a [3,1,P x S] array against the
//float [N,M,S] input array. Note that the dimensions must be checked before calling
//-- done by the wrapper -- and that only S elements of the appropriate row of
//the [3, 1, P x S] array are used.
template <typename T>
__global__ void conv1dDiagonalRademMultiply(T cArray[],
            const int8_t *rademArray,
			int dim2, int startPosition, int numElements,
            T normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        cArray[tid] = cArray[tid] * rademArray[position] * normConstant;
}




//We perform the transform over the last dimension
//of cArray which must be 3d; we expect cArray.shape[2] to 
//be a power of 2 (caller must verify). This can also be used
//for 2d arrays by passing dim1=1.
template <typename T>
void cudaHTransform(T cArray[],
		int dim0, int dim1, int dim2){

    int N, log2N;
    int spacing = 1;
    int arrsize = dim0 * dim1 * dim2;
    int blocksPerGrid;


    //baseLevelTransform only covers strides up to MAX_BASE_LEVEL_TRANSFORM.
    //If dim2 is less than that, we're set. Otherwise, run baseLevelTransform
    //first then use a somewhat slower global memory procedure for larger strides.
    N = (MAX_BASE_LEVEL_TRANSFORM < dim2) ? MAX_BASE_LEVEL_TRANSFORM : dim2;
    log2N = log2(N);
    blocksPerGrid = arrsize / N;

    baseLevelTransform<T><<<blocksPerGrid, N / 2, 
                    N * sizeof(T)>>>(cArray, N, log2N);
    
    if (dim2 <= MAX_BASE_LEVEL_TRANSFORM)
        return;
    
    //The largest strides (for large dim2) are handled by a somewhat
    //slower global memory procedure.
    spacing = MAX_BASE_LEVEL_TRANSFORM;
    blocksPerGrid = (arrsize / 2) + DEFAULT_THREADS_PER_BLOCK - 1;
    blocksPerGrid /= DEFAULT_THREADS_PER_BLOCK;
    while (spacing < dim2){
        levelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
        spacing <<= 1;
    }
}



//This function performs the SORF block transform (HD3 HD2 HD1) 
//Note that cArray must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria -- any other caller using this function
//should do the same.
//
//Note that all of these arrays are already expected to "live" on GPU.
//Can be used to perform SORF on a 2d array by passing dim1=1.
template <typename T>
const char *cudaSORF3d(T cArray[], int8_t *radem,
                int dim0, int dim1, int dim2){
    int numElementsPerRow = dim1 * dim2;
    int numElements = dim1 * dim2 * dim0;
    //This is the Hadamard normalization constant.
    T normConstant = log2(dim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    //hipProfilerStart();

    //Multiply by D1.
    diagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    cudaHTransform<T>(cArray, dim0, dim1, dim2);
    
    //Multiply by D2.
    diagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);

    //Second H-transform.
    cudaHTransform<T>(cArray, dim0, dim1, dim2);
    
    //Multiply by D3.
    diagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + 2 * numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);
    
    //Last H-transform. Transform is in place so do not need to return anything except no error message.
    cudaHTransform<T>(cArray, dim0, dim1, dim2); 

    //hipProfilerStop();
    return "no_error";
}
//Instantiate templates explicitly so wrapper can use.
template const char *cudaSORF3d<float>(float cArray[], int8_t *radem,
                int dim0, int dim1, int dim2);
template const char *cudaSORF3d<double>(double cArray[], int8_t *radem,
                int dim0, int dim1, int dim2);



//This function performs the SORF block transform (HD3 HD2 HD1)
//but for convolution-type operations.
//Note that cArray must have the same size across the
//last dimension as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria -- any other caller using this function
//should do the same.
//
//Note that all of these arrays are already expected to "live" on GPU.
template <typename T>
const char *cudaConvSORF3d(T cArray[], int8_t *radem,
                int dim0, int dim1, int dim2,
                int startPosition, int numElements,
                int rademShape2, T normConstant){
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    //hipProfilerStart();
        
    //Multiply by first row of radem.
    conv1dDiagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                        radem, dim2, startPosition, numElements,
                        normConstant);

    //First H-transform.
    cudaHTransform<T>(cArray, dim0, dim1, dim2);

    //Multiply by second row of radem.
    conv1dDiagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                        radem + rademShape2, dim2, startPosition, numElements,
                        normConstant);
    //Second H-transform.
    cudaHTransform<T>(cArray, dim0, dim1, dim2);
        
    //Multiply by third row of radem.
    conv1dDiagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray,
                        radem + 2 * rademShape2, dim2, startPosition, numElements,
                        normConstant);
    //Last H-transform.
    cudaHTransform<T>(cArray, dim0, dim1, dim2);

    //hipProfilerStop();
    return "no_error";
}
//Instantiate templates explicitly so wrapper can use.
template const char *cudaConvSORF3d<float>(float cArray[], int8_t *radem,
                int dim0, int dim1, int dim2, int startPosition,
                int numElements, int rademShape2, float normConstant);
template const char *cudaConvSORF3d<double>(double cArray[], int8_t *radem,
                int dim0, int dim1, int dim2, int startPosition,
                int numElements, int rademShape2, double normConstant);





//Performs the first two steps of SRHT (HD)
//Note that cArray must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria -- any other caller using this function
//should do the same.
//
//Note that all of these arrays are already expected to "live" on GPU.
template <typename T>
const char *cudaSRHT2d(T cArray[], int8_t *radem,
                int dim0, int dim1){
    int numElementsPerRow = dim1;
    int numElements = dim1 * dim0;
    //This is the Hadamard normalization constant.
    T normConstant = log2(dim1) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;

    //hipProfilerStart();

    //Multiply by D1.
    diagonalRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    cudaHTransform<T>(cArray, dim0, 1, dim1);

    //hipProfilerStop();
    return "no_error";
}
//Instantiate templates explicitly so wrapper can use.
template const char *cudaSRHT2d<float>(float cArray[], int8_t *radem,
                int dim0, int dim1);
template const char *cudaSRHT2d<double>(double cArray[], int8_t *radem,
                int dim0, int dim1);
