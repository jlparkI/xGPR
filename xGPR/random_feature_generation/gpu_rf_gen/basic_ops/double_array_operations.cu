#include "hip/hip_runtime.h"
/*
* Contains all functions needed to run the structured orthogonal features
* (SORF) and randomized Hadamard transform (RHT) operations on an input 3d
* array of doubles on GPU. Note that many operations here assume specific dimensions
* of the input array are a power of 2. The Cython wrapper checks this, so do
* not call these routines OUTSIDE of the Cython wrapper -- use the Cython wrapper.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "double_array_operations.h"
#include <hip/hip_runtime_api.h>

#define DEFAULT_THREADS_PER_BLOCK 256
#define MAX_BASE_LEVEL_TRANSFORM 512



//Uses shared memory to perform a reasonably efficient single kernel
//transform covering strides up to MAX_BASE_LEVEL_TRANSFORM.
__global__ void doubleBaseLevelTransform(double *cArray, int N, int log2N){
    int startPos = blockIdx.x << log2N;

    extern __shared__ double s_data[];
    int i, spacing, pos = threadIdx.x;
    int lo, id1, id2;
    double *src_ptr = cArray + startPos;
    double y;

    for (i = threadIdx.x; i < N; i += blockDim.x)
        s_data[i] = src_ptr[i];

    id1 = (pos << 1);
    id2 = id1 + 1;
    __syncthreads();
    y = s_data[id2];
    s_data[id2] = s_data[id1] - y;
    s_data[id1] += y;


    for (spacing = 2; spacing < N; spacing <<= 1){
        //Equivalent to pos mod spacing IF spacing is a power of 2.
        lo = pos & (spacing - 1);
        id1 = ((pos - lo) << 1) + lo;
        id2 = id1 + spacing;
        __syncthreads();
        y = s_data[id2];
        s_data[id2] = s_data[id1] - y;
        s_data[id1] += y;
    }
    for (i = threadIdx.x; i < N; i += blockDim.x){
        src_ptr[i] = s_data[i];
    }
}

//Combines the level 2 transform and the level 4 transform in
//global memory for arrays with small sizes where shape[2] < 32
//so that the shared memory procedure is not efficient.
__global__ void doubleShape4Transform(double *cArray, int arrsize)
{
    int id = 4 * (blockDim.x * blockIdx.x + threadIdx.x);
    int id1 = id, id2 = id + 1;
    double y;
    if (id < arrsize){
        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
        id1 += 2;
        id2 += 2;

        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
        id1 -= 1;

        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
        id1 -= 1;
        id2 -= 1;

        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
    }
}



//Performs subsequent stages of the transform (strides
//> MAX_BASE_LEVEL_TRANSFORM) using a less efficient global
//memory procedure.
__global__ void doubleLevelNTransform(double *cArray, int arrsize,
                                int spacing)
{
    int pos = blockDim.x * blockIdx.x + threadIdx.x;
    //Equivalent to pos mod spacing IF spacing is a power of 2.
    int lo = (pos & (spacing - 1));
    int id = lo + ((pos - lo) << 1);
    double y, *cPtr = cArray + id;
    
    if (id < arrsize){
        y = cPtr[spacing];
        cPtr[spacing] = *cPtr - y;
        *cPtr += y;
    }
}


//Performs an elementwise multiplication of a [c,M,P] array against the
//[N,M,P] input array or a [P] array against the [N,P] input array.
//Note that the last dimensions of these must be the
//same, and this function does not check this -- caller must check. Note that
//we mutiiply by the Hadamard normalization constant here.
__global__ void doubleMultiplyByDiagonalRademacherMat(double *cArray, int8_t *rademArray,
			int numElementsPerRow, int numElements, double normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rVal, position;
    
    position = tid % numElementsPerRow;
    rVal = rademArray[position];
    if (tid < numElements)
        cArray[tid] = cArray[tid] * rVal * normConstant;
}




//This function performs the SORF block transform (HD3 HD2 HD1) 
//Note that cArray must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria -- any other caller using this function
//should do the same.
//
//Note that all of these arrays are already expected to "live" on GPU.
const char *doubleCudaSORF3d(double *cArray, int8_t *radem,
                int dim0, int dim1, int dim2){
    int numElementsPerRow = dim1 * dim2;
    int numElements = dim1 * dim2 * dim0;
    //This is the Hadamard normalization constant.
    double normConstant = log2(dim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    //hipProfilerStart();

    //Multiply by D1.
    doubleMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    doubleCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D2.
    doubleMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);

    //Second H-transform.
    doubleCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D3.
    doubleMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + 2 * numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);
    
    //Last H-transform. Transform is in place so do not need to return anything except no error message.
    doubleCudaHTransform3d(cArray, dim0, dim1, dim2); 

    //hipProfilerStop();
    return "no_error";
}


//Performs the first two steps of SRHT (HD)
//Note that cArray must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria -- any other caller using this function
//should do the same.
//
//Note that all of these arrays are already expected to "live" on GPU.
const char *doubleCudaSRHT2d(double *cArray, int8_t *radem,
                int dim0, int dim1){
    int numElementsPerRow = dim1;
    int numElements = dim1 * dim0;
    //This is the Hadamard normalization constant.
    double normConstant = log2(dim1) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    //hipProfilerStart();

    //Multiply by D1.
    doubleMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    doubleCudaHTransform2d(cArray, dim0, dim1);

    //hipProfilerStop();
    return "no_error";
}



//We perform the transform over the last dimension
//of cArray which must be 3d; we expect cArray.shape[2] to 
//be a power of 2 (caller must verify).
void doubleCudaHTransform3d(double *cArray,
		int dim0, int dim1, int dim2){

    int N, log2N;
    int spacing = 1;
    int arrsize = dim0 * dim1 * dim2;
    int blocksPerGrid;

    //For less than 64, use specialized routines. dim2 is always
    //a power of two, and for best performance on CUDA threads per block
    //should be a multiple of 32, so the baseLevelTransform does
    //not work as well for dim2 < 64. There is a great deal of room
    //for additional optimization here that we have not done (yet) 
    //because input dim < 32 but > 2 is a somewhat niche application.
    if (dim2 < 32){
        blocksPerGrid = getNumBlocksDoubleTransform(arrsize, 2);
        if (dim2 == 2){
            doubleLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, 1);
        }
        else{
            doubleShape4Transform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize);
            spacing = 4;
            while (spacing < dim2){
                doubleLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
                spacing <<= 1;
            }
        }
        return;
    }

    //Otherwise, we use the baseLevelTransform, which uses shared
    //memory and is relatively efficient. baseLevelTransform only
    //covers strides up to MAX_BASE_LEVEL_TRANSFORM. If dim2 is less than that,
    //we're set. Otherwise, run baseLevelTransform first then use
    //a somewhat slower global memory procedure for larger strides.
    N = (MAX_BASE_LEVEL_TRANSFORM < dim2) ? MAX_BASE_LEVEL_TRANSFORM : dim2;
    log2N = log2(N);
    blocksPerGrid = arrsize / N;

    doubleBaseLevelTransform<<<blocksPerGrid, N / 2, 
                    N * sizeof(double)>>>(cArray, N, log2N);
    
    if (dim2 <= MAX_BASE_LEVEL_TRANSFORM)
        return;
    
    //The largest strides (for large dim2) are handled by a somewhat
    //slower global memory procedure.
    spacing = MAX_BASE_LEVEL_TRANSFORM;
    blocksPerGrid = getNumBlocksDoubleTransform(arrsize, 2);
    while (spacing < dim2){
        doubleLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
        spacing <<= 1;
    }
}


//We perform the transform over the last dimension
//of cArray which must be 2d; we expect cArray.shape[1] to 
//be a power of 2 (caller must verify).
void doubleCudaHTransform2d(double *cArray, int dim0, int dim1){

    int N, log2N;
    int spacing = 1;
    int arrsize = dim0 * dim1;
    int blocksPerGrid;

    //For less than 64, use specialized routines. dim2 is always
    //a power of two, and for best performance on CUDA threads per block
    //should be a multiple of 32, so the baseLevelTransform does
    //not work as well for dim2 < 64. There is a great deal of room
    //for additional optimization here that we have not done (yet) 
    //because input dim < 32 but > 2 is a somewhat niche application.
    if (dim1 < 32){
        blocksPerGrid = getNumBlocksDoubleTransform(arrsize, 2);
        if (dim1 == 2){
            doubleLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, 1);
        }
        else{
            doubleShape4Transform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize);
            spacing = 4;
            while (spacing < dim1){
                doubleLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
                spacing <<= 1;
            }
        }
        return;
    }

    //Otherwise, we use the baseLevelTransform, which uses shared
    //memory and is relatively efficient. baseLevelTransform only
    //covers strides up to MAX_BASE_LEVEL_TRANSFORM. If dim2 is less than that,
    //we're set. Otherwise, run baseLevelTransform first then use
    //a somewhat slower global memory procedure for larger strides.
    N = (MAX_BASE_LEVEL_TRANSFORM < dim1) ? MAX_BASE_LEVEL_TRANSFORM : dim1;
    log2N = log2(N);
    blocksPerGrid = arrsize / N;

    doubleBaseLevelTransform<<<blocksPerGrid, N / 2, 
                    N * sizeof(double)>>>(cArray, N, log2N);
    
    if (dim1 <= MAX_BASE_LEVEL_TRANSFORM)
        return;
    
    //The largest strides (for large dim1) are handled by a somewhat
    //slower global memory procedure.
    spacing = MAX_BASE_LEVEL_TRANSFORM;
    blocksPerGrid = getNumBlocksDoubleTransform(arrsize, 2);
    while (spacing < dim1){
        doubleLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
        spacing <<= 1;
    }
}





//Calculates the number of blocks for all transforms except the 
//baseLevelTransform, which uses shared memory and hence
//a slightly different procedure.
int getNumBlocksDoubleTransform(int arrsize, int divisor){

    int blocksPerGrid;
    blocksPerGrid = (arrsize / divisor) + DEFAULT_THREADS_PER_BLOCK - 1;
    return blocksPerGrid / DEFAULT_THREADS_PER_BLOCK;
}
