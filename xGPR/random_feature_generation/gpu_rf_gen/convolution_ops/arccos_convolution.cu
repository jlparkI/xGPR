#include "hip/hip_runtime.h"
/*
* Contains routines needed specifically for generating features for ArcCos-based
* convolution kernels and calculating their gradients.
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../basic_ops/float_array_operations.h"
#include "../basic_ops/double_array_operations.h"
#include "arccos_convolution.h"

#define DEFAULT_THREADS_PER_BLOCK 256
#define DEFAULT_THREADS_PER_BLREDUCE 32




//Performs an elementwise multiplication of a row of a [3,1,P x S] array against the
//float [N,M,S] input array. Note that the dimensions must be checked before calling
//-- done by the wrapper -- and that only S elements of the appropriate row of
//the [3, 1, P x S] array are used.
__global__ void floatConv1dArcCosRademMultiply(float *cArray,
            const int8_t *rademArray,
			int dim2, int startPosition, int numElements, float normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        cArray[tid] = cArray[tid] * rademArray[position] * normConstant;
}



//Performs an elementwise multiplication of a row of a [3,1,P x S] array against the
//double [N,M,S] input array. Note that the dimensions must be checked before calling
//-- done by the wrapper -- and that only S elements of the appropriate row of
//the [3, 1, P x S] array are used.
__global__ void doubleConv1dArcCosRademMultiply(double *cArray,
            const int8_t *rademArray,
			int dim2, int startPosition, int numElements, double normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        cArray[tid] = cArray[tid] * rademArray[position] * normConstant;
}



//Performs an elementwise multiplication by a diagonal matrix populated with
//elements from a Rademacher distribution, while also multiplying by the
//Hadamard norm constant and copying into the featureArray array.
__global__ void floatConv1dArcCosRademAndCopy(const float *inputArray, float *featureArray,
            const int8_t *rademArray, int dim2, int startPosition,
            int numElements, float normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        featureArray[tid] = inputArray[tid] * rademArray[position] * normConstant;
}


//Performs an elementwise multiplication by a diagonal matrix populated with
//elements from a Rademacher distribution, while also multiplying by the
//Hadamard norm constant and copying into the featureArray array.
__global__ void doubleConv1dArcCosRademAndCopy(const double *inputArray, double *featureArray,
            const int8_t *rademArray, int dim2, int startPosition,
            int numElements, double normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        featureArray[tid] = inputArray[tid] * rademArray[position] * normConstant;
}




//Performs the final steps in feature generation for ArcCos-based convolution
//kernels -- multiplying by chiArr, taking sine or cosine and adding
//to the appropriate elements of outputArray.
__global__ void floatConvArcCosPostProcessKernelOrder1(const float *featureArray, float *chiArr,
            double *outputArray, int dim1, int dim2, int numFreqs,
            int startPosition, int numElements,
            int endPosition, double scalingTerm){
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int column = tid % endPosition;
    int row = tid / endPosition;
    int inputLoc = row * dim1 * dim2 + column;
    int outputLoc = row * numFreqs + column + startPosition;
    float *chiVal = chiArr + startPosition + column;
    float chiProd, rollingSum = 0;

    if (tid < numElements){
        for (i=0; i < dim1; i++){
            chiProd = *chiVal * featureArray[inputLoc];
            rollingSum += max(chiProd, 0.0);
            inputLoc += dim2;
        }
        outputArray[outputLoc] = rollingSum * scalingTerm;
    }
}



//Performs the final steps in feature generation for ArcCos-based convolution
//kernels -- multiplying by chiArr, taking sine or cosine and adding
//to the appropriate elements of outputArray.
__global__ void doubleConvArcCosPostProcessKernelOrder1(const double *featureArray, double *chiArr,
            double *outputArray, int dim1, int dim2, int numFreqs,
            int startPosition, int numElements,
            int endPosition, double scalingTerm)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int column = tid % endPosition;
    int row = tid / endPosition;
    int inputLoc = row * dim1 * dim2 + column;
    int outputLoc = row * numFreqs + column + startPosition;
    double *chiVal = chiArr + startPosition + column;
    double chiProd, rollingSum = 0;

    if (tid < numElements){
        for (i=0; i < dim1; i++){
            chiProd = *chiVal * featureArray[inputLoc];
            rollingSum += max(chiProd, 0.0);
            inputLoc += dim2;
        }
        outputArray[outputLoc] = rollingSum * scalingTerm;
    }
}



//Performs the final steps in feature generation for ArcCos-based convolution
//kernels -- multiplying by chiArr, taking sine or cosine and adding
//to the appropriate elements of outputArray.
__global__ void floatConvArcCosPostProcessKernelOrder2(const float *featureArray, float *chiArr,
            double *outputArray, int dim1, int dim2, int numFreqs,
            int startPosition, int numElements,
            int endPosition, double scalingTerm){
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int column = tid % endPosition;
    int row = tid / endPosition;
    int inputLoc = row * dim1 * dim2 + column;
    int outputLoc = row * numFreqs + column + startPosition;
    float *chiVal = chiArr + startPosition + column;
    float chiProd, rollingSum = 0;

    if (tid < numElements){
        for (i=0; i < dim1; i++){
            chiProd = *chiVal * featureArray[inputLoc];
            chiProd = max(chiProd, 0.0);
            rollingSum += chiProd * chiProd;
            inputLoc += dim2;
        }
        outputArray[outputLoc] = rollingSum * scalingTerm;
    }
}



//Performs the final steps in feature generation for ArcCos-based convolution
//kernels -- multiplying by chiArr, taking sine or cosine and adding
//to the appropriate elements of outputArray.
__global__ void doubleConvArcCosPostProcessKernelOrder2(const double *featureArray, double *chiArr,
            double *outputArray, int dim1, int dim2, int numFreqs,
            int startPosition, int numElements,
            int endPosition, double scalingTerm)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int column = tid % endPosition;
    int row = tid / endPosition;
    int inputLoc = row * dim1 * dim2 + column;
    int outputLoc = row * numFreqs + column + startPosition;
    double *chiVal = chiArr + startPosition + column;
    double chiProd, rollingSum = 0;

    if (tid < numElements){
        for (i=0; i < dim1; i++){
            chiProd = *chiVal * featureArray[inputLoc];
            chiProd = max(chiProd, 0.0);
            rollingSum += chiProd * chiProd;
            inputLoc += dim2;
        }
        outputArray[outputLoc] = rollingSum * scalingTerm;
    }
}



//This function generates and sums random features for an
//input array reshapedX of input type float.
const char *floatConvArcCosFeatureGen(int8_t *radem, float *reshapedX,
            float *featureArray, float *chiArr, double *outputArray,     
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm,
            int kernelOrder){

    int numElements = reshapedDim0 * reshapedDim1 * reshapedDim2;
    //This is the Hadamard normalization constant.
    float normConstant = log2(reshapedDim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / 
                DEFAULT_THREADS_PER_BLOCK;

    int endPosition, numOutElements, outBlocks;
    int numRepeats = (numFreqs + reshapedDim2 - 1) / reshapedDim2;
    int i, startPosition;

    for (i=0; i < numRepeats; i++){
        startPosition = i * reshapedDim2;
        endPosition = MIN((i + 1) * reshapedDim2, numFreqs);
        endPosition -= i * reshapedDim2;
        numOutElements = reshapedDim0 * endPosition;
        outBlocks = (numOutElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;

        //Copy input into featureArray while multiplying by first row of radem.
        floatConv1dArcCosRademAndCopy<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        featureArray, radem, reshapedDim2, startPosition, numElements,
                        normConstant);
        //First H-transform.
        floatCudaHTransform3d(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);

        //Multiply by second row of radem.
        floatConv1dArcCosRademMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Second H-transform.
        floatCudaHTransform3d(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);
        
        //Multiply by third row of radem.
        floatConv1dArcCosRademMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + 2 * rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Last H-transform.
        floatCudaHTransform3d(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);


        //Multiply by chiArr; take the sine and cosine of elements of
        //featureArray, multiply by scalingTerm, and transfer to outputArray.
        if (kernelOrder == 1){
            floatConvArcCosPostProcessKernelOrder1<<<outBlocks, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, chiArr,
                outputArray, reshapedDim1, reshapedDim2, numFreqs,
                startPosition, numOutElements, endPosition,
                scalingTerm);
        }
        else{
            floatConvArcCosPostProcessKernelOrder2<<<outBlocks, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, chiArr,
                outputArray, reshapedDim1, reshapedDim2, numFreqs,
                startPosition, numOutElements, endPosition,
                scalingTerm);
        }
    }

    return "no_error";
}



//This function generates and sums random features for an
//input array reshapedX of input type double.
const char *doubleConvArcCosFeatureGen(int8_t *radem, double *reshapedX,
                double *featureArray, double *chiArr, double *outputArray,
                int reshapedDim0, int reshapedDim1, int reshapedDim2,
                int numFreqs, int rademShape2, double scalingTerm,
                int kernelOrder){

    int numElements = reshapedDim0 * reshapedDim1 * reshapedDim2;
    //This is the Hadamard normalization constant.
    double normConstant = log2(reshapedDim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / 
                DEFAULT_THREADS_PER_BLOCK;

    int endPosition, numOutElements, outBlocks;
    int numRepeats = (numFreqs + reshapedDim2 - 1) / reshapedDim2;
    int i, startPosition;

    for (i=0; i < numRepeats; i++){
        startPosition = i * reshapedDim2;
        endPosition = MIN((i + 1) * reshapedDim2, numFreqs);
        endPosition -= i * reshapedDim2;
        numOutElements = reshapedDim0 * endPosition;
        outBlocks = (numOutElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    
        //Copy input into featureArray while multiplying by first row of radem.
        doubleConv1dArcCosRademAndCopy<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        featureArray, radem, reshapedDim2, startPosition, numElements,
                        normConstant);
        //First H-transform.
        doubleCudaHTransform3d(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);

        //Multiply by second row of radem.
        doubleConv1dArcCosRademMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Second H-transform.
        doubleCudaHTransform3d(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);
        
        //Multiply by third row of radem.
        doubleConv1dArcCosRademMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + 2 * rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Last H-transform.
        doubleCudaHTransform3d(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);

        //Multiply by chiArr; take the sine and cosine of elements of
        //featureArray, multiply by scalingTerm, and transfer to output.
        if (kernelOrder == 1){
            doubleConvArcCosPostProcessKernelOrder1<<<outBlocks, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, chiArr,
                outputArray, reshapedDim1, reshapedDim2, numFreqs,
                startPosition, numOutElements, endPosition,
                scalingTerm);
        }
        else{
            doubleConvArcCosPostProcessKernelOrder2<<<outBlocks, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, chiArr,
                outputArray, reshapedDim1, reshapedDim2, numFreqs,
                startPosition, numOutElements, endPosition,
                scalingTerm);
        }
    }

    return "no_error";
}
