#include "hip/hip_runtime.h"
/*
* Contains routines needed specifically for generating features for RBF-based
* convolution kernels (FHTConv1d, GraphConv) and calculating their gradients.
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../basic_ops/basic_array_operations.h"
#include "rbf_convolution.h"

#define DEFAULT_THREADS_PER_BLOCK 256
#define DEFAULT_THREADS_PER_BLREDUCE 32




//Performs an elementwise multiplication of a row of a [3,1,P x S] array against the
//float [N,M,S] input array. Note that the dimensions must be checked before calling
//-- done by the wrapper -- and that only S elements of the appropriate row of
//the [3, 1, P x S] array are used.
template <typename T>
__global__ void conv1dRBFRademMultiply(T cArray[],
            const int8_t *rademArray,
			int dim2, int startPosition, int numElements,
            T normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        cArray[tid] = cArray[tid] * rademArray[position] * normConstant;
}

//Performs an elementwise multiplication by a diagonal matrix populated with
//elements from a Rademacher distribution, while also multiplying by the
//Hadamard norm constant and copying into the featureArray array.
template <typename T>
__global__ void conv1dRBFRademAndCopy(T inputArray[], T featureArray[],
            const int8_t *rademArray, int dim2, int startPosition,
            int numElements, T normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int position = startPosition + (tid & (dim2 - 1));
    
    if (tid < numElements)
        featureArray[tid] = inputArray[tid] * rademArray[position] * normConstant;
}




//Performs the final steps in feature generation for RBF-based convolution
//kernels -- multiplying by chiArr, taking sine or cosine and adding
//to the appropriate elements of outputArray.
template <typename T>
__global__ void convRBFPostProcessKernel(T featureArray[], T chiArr[],
            double *outputArray, int dim1, int dim2, int numFreqs,
            int startPosition, int numElements,
            int endPosition, double scalingTerm){
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int column = tid % endPosition;
    int row = tid / endPosition;
    int inputLoc = row * dim1 * dim2 + column;
    int outputLoc = row * 2 * numFreqs + 2 * column + 2 * startPosition;
    T chiVal = chiArr[startPosition + column];
    double chiProd, sinSum = 0, cosSum = 0;

    if (tid < numElements){
        for (i=0; i < dim1; i++){
            chiProd = chiVal * featureArray[inputLoc];
            cosSum += cos(chiProd);
            sinSum += sin(chiProd);
            inputLoc += dim2;
        }
        outputArray[outputLoc] = cosSum * scalingTerm;
        outputArray[outputLoc + 1] = sinSum * scalingTerm;
    }
}


//Performs the final steps in feature generation WITH simultaneous gradient
//calculation for RBF-based convolution
//kernels -- multiplying by chiArr, taking sine or cosine and adding
//to the appropriate elements of outputArray.
template <typename T>
__global__ void convRBFGradProcessKernel(T featureArray[], T chiArr[],
            double *outputArray, int dim1, int dim2, int numFreqs,
            int startPosition, int numElements,
            int endPosition, double scalingTerm,
            double sigma, double *gradientArray)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int column = tid % endPosition;
    int row = tid / endPosition;
    int inputLoc = row * dim1 * dim2 + column;
    int outputLoc = row * 2 * numFreqs + 2 * column + 2 * startPosition;
    T chiVal = chiArr[startPosition + column];
    double chiProd, sinSum = 0, cosSum = 0, sinVal, cosVal;
    double gradSinVal = 0, gradCosVal = 0;

    if (tid < numElements){
        for (i=0; i < dim1; i++){
            chiProd = chiVal * featureArray[inputLoc];
            cosVal = cos(chiProd * sigma);
            sinVal = sin(chiProd * sigma);

            cosSum += cosVal;
            sinSum += sinVal;
            //These are the derivatives.
            gradCosVal -= sinVal * chiProd;
            gradSinVal += cosVal * chiProd;
            inputLoc += dim2;
        }
        outputArray[outputLoc] = cosSum * scalingTerm;
        outputArray[outputLoc + 1] = sinSum * scalingTerm;

        gradientArray[outputLoc] = gradCosVal * scalingTerm;
        gradientArray[outputLoc + 1] = gradSinVal * scalingTerm;
    }
}



//This function generates and sums random features for an
//input array reshapedX of input type float.
template <typename T>
const char *convRBFFeatureGen(int8_t *radem, T reshapedX[],
            T featureArray[], T chiArr[], double *outputArray,     
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm){

    int numElements = reshapedDim0 * reshapedDim1 * reshapedDim2;
    //This is the Hadamard normalization constant.
    T normConstant = log2(reshapedDim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / 
                DEFAULT_THREADS_PER_BLOCK;

    int endPosition, numOutElements, outBlocks;
    int numRepeats = (numFreqs + reshapedDim2 - 1) / reshapedDim2;
    int i, startPosition;

    for (i=0; i < numRepeats; i++){
        startPosition = i * reshapedDim2;
        endPosition = MIN((i + 1) * reshapedDim2, numFreqs);
        endPosition -= i * reshapedDim2;
        numOutElements = reshapedDim0 * endPosition;
        outBlocks = (numOutElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;

        //Copy input into featureArray while multiplying by first row of radem.
        conv1dRBFRademAndCopy<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        featureArray, radem, reshapedDim2, startPosition, numElements,
                        normConstant);
        //First H-transform.
        cudaHTransform3d<T>(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);

        //Multiply by second row of radem.
        conv1dRBFRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Second H-transform.
        cudaHTransform3d<T>(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);
        
        //Multiply by third row of radem.
        conv1dRBFRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + 2 * rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Last H-transform.
        cudaHTransform3d<T>(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);


        //Multiply by chiArr; take the sine and cosine of elements of
        //featureArray, multiply by scalingTerm, and transfer to outputArray.
        convRBFPostProcessKernel<T><<<outBlocks, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, chiArr,
                outputArray, reshapedDim1, reshapedDim2, numFreqs,
                startPosition, numOutElements, endPosition,
                scalingTerm);
    }

    return "no_error";
}
//Explicitly instantiate so wrapper can use.
template const char *convRBFFeatureGen<float>(int8_t *radem, float reshapedX[],
            float featureArray[], float chiArr[], double *outputArray,     
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm);
template const char *convRBFFeatureGen<double>(int8_t *radem, double reshapedX[],
            double featureArray[], double chiArr[], double *outputArray,     
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm);



//This function generates and sums random features for an
//input array reshapedX of input type float WHILE also
//generating gradient information and storing this in
//a separate array. This gradient is only applicable
//in cases where all of the features share the same
//lengthscale; ARD-type kernels require a more complicated
//gradient calculation not implemented here.
template <typename T>
const char *convRBFFeatureGrad(int8_t *radem, T reshapedX[],
            T featureArray[], T chiArr[], double *outputArray,     
            double *gradientArray, double sigma,
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm){

    int numElements = reshapedDim0 * reshapedDim1 * reshapedDim2;
    //This is the Hadamard normalization constant.
    T normConstant = log2(reshapedDim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / 
                DEFAULT_THREADS_PER_BLOCK;
    int numOutElements, outBlocks;

    int numRepeats = (numFreqs + reshapedDim2 - 1) / reshapedDim2;
    int i, startPosition, endPosition;

    for (i=0; i < numRepeats; i++){
        startPosition = i * reshapedDim2;
        endPosition = MIN((i + 1) * reshapedDim2, numFreqs);
        endPosition -= startPosition;
        numOutElements = reshapedDim0 * endPosition;
        outBlocks = (numOutElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;

        //Copy input into featureArray while multiplying by first row of radem.
        conv1dRBFRademAndCopy<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(reshapedX, 
                        featureArray, radem, reshapedDim2, startPosition, numElements,
                        normConstant);
        //First H-transform.
        cudaHTransform3d<T>(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);

        //Multiply by second row of radem.
        conv1dRBFRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Second H-transform.
        cudaHTransform3d<T>(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);
        
        //Multiply by third row of radem.
        conv1dRBFRademMultiply<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, 
                        radem + 2 * rademShape2, reshapedDim2, startPosition, numElements,
                        normConstant);
        //Last H-transform.
        cudaHTransform3d<T>(featureArray, reshapedDim0, reshapedDim1, reshapedDim2);

        //Multiply by chiArr; take the sine and cosine of elements of
        //featureArray, multiply by scalingTerm, transfer to output
        //AND at the same time calculate the gradient terms, using
        //them to populate gradientArray.
        convRBFGradProcessKernel<T><<<outBlocks, DEFAULT_THREADS_PER_BLOCK>>>(featureArray, chiArr,
                outputArray, reshapedDim1, reshapedDim2, numFreqs,
                startPosition, numOutElements, endPosition,
                scalingTerm, sigma, gradientArray);
    }

    return "no_error";
}
//Explicitly instantiate so wrapper can use.
template const char *convRBFFeatureGrad<float>(int8_t *radem, float reshapedX[],
            float featureArray[], float chiArr[], double *outputArray,     
            double *gradientArray, double sigma,
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm);
template const char *convRBFFeatureGrad<double>(int8_t *radem, double reshapedX[],
            double featureArray[], double chiArr[], double *outputArray,     
            double *gradientArray, double sigma,
            int reshapedDim0, int reshapedDim1, int reshapedDim2,
            int numFreqs, int rademShape2, double scalingTerm);
