#include "hip/hip_runtime.h"
/*
* Contains routines needed specifically for generating features for RBF-based
* convolution kernels (FHTConv1d, GraphConv) and calculating their gradients.
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../shared_constants.h"
#include "../sharedmem.h"
#include "rbf_convolution.h"

//Generates the Conv kernel RBF features. This single kernel loops over 1) kmers
//then 2) the number of repeats then inside that loop 3) the three diagonal
//matrix multiplications and fast Hadamard transforms before
//applying 4) diagonal matmul before activation function.
template <typename T>
__global__ void convRBFFeatureGenKernel(const T origData[], T cArray[],
        double *outputArray, const T chiArr[], const int8_t *radem,
        int paddedBufferSize, int log2N, int numFreqs, int xDim1, int xDim2,
        int nRepeats, int rademShape2, T normConstant,
        double scalingConstant, int scalingType,
        int convWidth, const int32_t *seqlengths){

    int stepSize = MIN(paddedBufferSize, MAX_BASE_LEVEL_TRANSFORM);
    int colCutoff = seqlengths[blockIdx.x] - convWidth + 1;

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int spacing, pos = threadIdx.x;
    int lo, id1, id2;
    int tempArrPos, chiArrPos = 0, inputCutoff = xDim2 * convWidth;
    int inputArrPos = (blockIdx.x * xDim1 * xDim2);
    int outputArrPos = (blockIdx.x * numFreqs * 2);
    T y, outputVal, modifiedScaling = scalingConstant;

    const int8_t *rademPtr = radem;

    switch (scalingType){
        case 0:
            break;
        case 1:
            modifiedScaling = modifiedScaling / sqrt( (double) colCutoff);
            break;
        case 2:
            modifiedScaling = modifiedScaling / (double) colCutoff;
            break;
    }

    //Loop over the kmers in this stretch.
    for (int kmer = 0; kmer < colCutoff; kmer++){
        chiArrPos = 0;
        outputArrPos = (blockIdx.x * numFreqs * 2);
        inputArrPos = (blockIdx.x * xDim1 * xDim2) + kmer * xDim2;

        //Run over the number of repeats required to generate the random
        //features.
        for (int rep = 0; rep < nRepeats; rep++){
            tempArrPos = (blockIdx.x << log2N);

            //Copy original data into the temporary array.
            for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
                if (i < inputCutoff)
                    cArray[i + tempArrPos] = origData[i + inputArrPos];
                else
                    cArray[i + tempArrPos] = 0;
            }

            //Run over three repeats for the SORF procedure.
            for (int sorfRep = 0; sorfRep < 3; sorfRep++){
                rademPtr = radem + paddedBufferSize * rep + sorfRep * rademShape2;
                tempArrPos = (blockIdx.x << log2N);

                for (int hStep = 0; hStep < paddedBufferSize; hStep+=stepSize){
                    for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                        s_data[i] = cArray[i + tempArrPos];

                    __syncthreads();

                    //Multiply by the diagonal array here.
                    for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                        s_data[i] = s_data[i] * rademPtr[i] * normConstant;

                    rademPtr += stepSize;

                    id1 = (pos << 1);
                    id2 = id1 + 1;
                    __syncthreads();
                    y = s_data[id2];
                    s_data[id2] = s_data[id1] - y;
                    s_data[id1] += y;

                    for (spacing = 2; spacing < stepSize; spacing <<= 1){
                        //Equivalent to pos mod spacing if spacing is a power of 2,
                        //which here is always true.
                        lo = pos & (spacing - 1);
                        id1 = ((pos - lo) << 1) + lo;
                        id2 = id1 + spacing;
                        __syncthreads();
                        y = s_data[id2];
                        s_data[id2] = s_data[id1] - y;
                        s_data[id1] += y;
                    }
                    __syncthreads();

                    for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                        cArray[i + tempArrPos] = s_data[i];

                    tempArrPos += stepSize;
                    __syncthreads();
                }

                //A less efficient global memory procedure to complete the FHT
                //for long arrays.
                if (paddedBufferSize > MAX_BASE_LEVEL_TRANSFORM){
                    tempArrPos = (blockIdx.x << log2N);

                    for (int spacing = stepSize; spacing < paddedBufferSize; spacing <<= 1){

                        for (int k = 0; k < paddedBufferSize; k += (spacing << 1)){
                            for (int i = threadIdx.x; i < spacing; i += blockDim.x){
                                id1 = i + k + tempArrPos;
                                id2 = id1 + spacing;
                                y = cArray[id2];
                                cArray[id2] = cArray[id1] - y;
                                cArray[id1] += y;
                            }
                            __syncthreads();
                        }
                    }
                }
            }
            //Now take the results stored in the temporary array, apply the
            //activation function, and populate the output array. Note that
            //we multiply by 2 in the output array position since two
            //features are generated for each frequency sampled.
            tempArrPos = (blockIdx.x << log2N);

            for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
                if ((i + chiArrPos) >= numFreqs)
                    break;
                outputVal = chiArr[chiArrPos + i] * cArray[tempArrPos + i];
                outputArray[outputArrPos + 2 * i] += modifiedScaling * cos(outputVal);
                outputArray[outputArrPos + 2 * i + 1] += modifiedScaling * sin(outputVal);
            }

            chiArrPos += paddedBufferSize;
            outputArrPos += 2 * paddedBufferSize;
            __syncthreads();
        }
    }
}




//Generates the Conv kernel RBF features together with the gradient. This single
//kernel loops over 1) kmers then 2) the number of repeats then inside that
//loop 3) the three diagonal matrix multiplications and fast Hadamard transforms
//before applying 4) diagonal matmul before activation function.
template <typename T>
__global__ void convRBFFeatureGradKernel(const T origData[], T cArray[],
        double *outputArray, const T chiArr[], const int8_t *radem,
        int paddedBufferSize, int log2N, int numFreqs, int xDim1, int xDim2,
        int nRepeats, int rademShape2, T normConstant,
        double scalingConstant, int scalingType,
        int convWidth, const int32_t *seqlengths,
        double *gradient, T sigma){

    int stepSize = MIN(paddedBufferSize, MAX_BASE_LEVEL_TRANSFORM);
    int colCutoff = seqlengths[blockIdx.x] - convWidth + 1;

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int spacing, pos = threadIdx.x;
    int lo, id1, id2;
    int tempArrPos, chiArrPos = 0, inputCutoff = xDim2 * convWidth;
    int inputArrPos = (blockIdx.x * xDim1 * xDim2);
    int outputArrPos = (blockIdx.x * numFreqs * 2);
    T y, outputVal, modifiedScaling = scalingConstant;

    const int8_t *rademPtr = radem;

    switch (scalingType){
        case 0:
            break;
        case 1:
            modifiedScaling = modifiedScaling / sqrt( (double) colCutoff);
            break;
        case 2:
            modifiedScaling = modifiedScaling / (double) colCutoff;
            break;
    }

    //Loop over the kmers in this stretch.
    for (int kmer = 0; kmer < colCutoff; kmer++){
        chiArrPos = 0;
        outputArrPos = (blockIdx.x * numFreqs * 2);
        inputArrPos = (blockIdx.x * xDim1 * xDim2) + kmer * xDim2;

        //Run over the number of repeats required to generate the random
        //features.
        for (int rep = 0; rep < nRepeats; rep++){
            tempArrPos = (blockIdx.x << log2N);

            //Copy original data into the temporary array.
            for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
                if (i < inputCutoff)
                    cArray[i + tempArrPos] = origData[i + inputArrPos];
                else
                    cArray[i + tempArrPos] = 0;
            }

            //Run over three repeats for the SORF procedure.
            for (int sorfRep = 0; sorfRep < 3; sorfRep++){
                rademPtr = radem + paddedBufferSize * rep + sorfRep * rademShape2;
                tempArrPos = (blockIdx.x << log2N);

                for (int hStep = 0; hStep < paddedBufferSize; hStep+=stepSize){
                    for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                        s_data[i] = cArray[i + tempArrPos];

                    __syncthreads();

                    //Multiply by the diagonal array here.
                    for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                        s_data[i] = s_data[i] * rademPtr[i] * normConstant;

                    rademPtr += stepSize;

                    id1 = (pos << 1);
                    id2 = id1 + 1;
                    __syncthreads();
                    y = s_data[id2];
                    s_data[id2] = s_data[id1] - y;
                    s_data[id1] += y;

                    for (spacing = 2; spacing < stepSize; spacing <<= 1){
                        //Equivalent to pos mod spacing if spacing is a power of 2,
                        //which here is always true.
                        lo = pos & (spacing - 1);
                        id1 = ((pos - lo) << 1) + lo;
                        id2 = id1 + spacing;
                        __syncthreads();
                        y = s_data[id2];
                        s_data[id2] = s_data[id1] - y;
                        s_data[id1] += y;
                    }
                    __syncthreads();

                    for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                        cArray[i + tempArrPos] = s_data[i];

                    tempArrPos += stepSize;
                    __syncthreads();
                }

                //A less efficient global memory procedure to complete the FHT
                //for long arrays.
                if (paddedBufferSize > MAX_BASE_LEVEL_TRANSFORM){
                    tempArrPos = (blockIdx.x << log2N);

                    for (int spacing = stepSize; spacing < paddedBufferSize; spacing <<= 1){

                        for (int k = 0; k < paddedBufferSize; k += (spacing << 1)){
                            for (int i = threadIdx.x; i < spacing; i += blockDim.x){
                                id1 = i + k + tempArrPos;
                                id2 = id1 + spacing;
                                y = cArray[id2];
                                cArray[id2] = cArray[id1] - y;
                                cArray[id1] += y;
                            }
                            __syncthreads();
                        }
                    }
                }
            }
            //Now take the results stored in the temporary array, apply the
            //activation function, and populate the output array. Note that
            //we multiply by 2 in the output array position since two
            //features are generated for each frequency sampled.
            tempArrPos = (blockIdx.x << log2N);

            for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
                if ((i + chiArrPos) >= numFreqs)
                    break;
                outputVal = chiArr[chiArrPos + i] * cArray[tempArrPos + i];
                double prodVal = outputVal * sigma;
                outputArray[outputArrPos + 2 * i] += modifiedScaling * cos(prodVal);
                outputArray[outputArrPos + 2 * i + 1] += modifiedScaling * sin(prodVal);
                gradient[outputArrPos + 2 * i] -= modifiedScaling * sin(prodVal) * outputVal;
                gradient[outputArrPos + 2 * i + 1] += modifiedScaling * cos(prodVal) * outputVal;
            }

            chiArrPos += paddedBufferSize;
            outputArrPos += 2 * paddedBufferSize;
            __syncthreads();
        }
    }
}






//This function generates and sums random features for a Conv1d RBF-type kernel.
template <typename T>
int convRBFFeatureGen(nb::ndarray<T, nb::shape<-1,-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<int8_t, nb::shape<3, 1, -1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<T, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        nb::ndarray<int32_t, nb::shape<-1>, nb::device::cpu, nb::c_contig> seqlengths,
        int convWidth, int scalingType) {

    // Perform safety checks. Any exceptions thrown here are handed off to Python
    // by the Nanobind wrapper. We do not expect the user to see these because
    // the Python code will always ensure inputs are correct -- these are a failsafe
    // -- so we do not need to provide detailed exception messages here.
    int zDim0 = inputArr.shape(0);
    int zDim1 = inputArr.shape(1);
    int zDim2 = inputArr.shape(2);
    size_t numRffs = outputArr.shape(1);
    size_t numFreqs = chiArr.shape(0);
    double scalingTerm = std::sqrt(1.0 / static_cast<double>(numFreqs));

    T *inputPtr = inputArr.data();
    double *outputPtr = outputArr.data();
    T *chiPtr = chiArr.data();
    int8_t *rademPtr = radem.data();
    int32_t *seqlengthsPtr = seqlengths.data();

    if (inputArr.shape(0) == 0 || outputArr.shape(0) != inputArr.shape(0))
        throw std::runtime_error("no datapoints");
    if (numRffs < 2 || (numRffs & 1) != 0)
        throw std::runtime_error("last dim of output must be even number");
    if ( (2 * numFreqs) != numRffs || numFreqs > radem.shape(2) )
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    if (seqlengths.shape(0) != inputArr.shape(0))
        throw std::runtime_error("wrong array sizes");
    if (static_cast<int>(inputArr.shape(1)) < convWidth || convWidth <= 0)
        throw std::runtime_error("invalid conv_width");

    double expectedNFreq = static_cast<double>(convWidth * inputArr.shape(2));
    expectedNFreq = MAX(expectedNFreq, 2);
    double log2Freqs = std::log2(expectedNFreq);
    log2Freqs = std::ceil(log2Freqs);
    int paddedBufferSize = std::pow(2, log2Freqs);

    if (radem.shape(2) % paddedBufferSize != 0)
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    int32_t minSeqLength = 2147483647, maxSeqLength = 0;
    for (size_t i=0; i < seqlengths.shape(0); i++){
        if (seqlengths(i) > maxSeqLength)
            maxSeqLength = seqlengths(i);
        if (seqlengths(i) < minSeqLength)
            minSeqLength = seqlengths(i);
    }

    if (maxSeqLength > static_cast<int32_t>(inputArr.shape(1)) || minSeqLength < convWidth){
        throw std::runtime_error("All sequence lengths must be >= conv width and < "
                "array size.");
    }

    int32_t *slenCudaPtr;
    if (hipMalloc(&slenCudaPtr, sizeof(int32_t) * seqlengths.shape(0)) != hipSuccess) {
        hipFree(slenCudaPtr);
        throw std::runtime_error("Cuda is out of memory");
        return 1;
    };
    if (hipMemcpy(slenCudaPtr, seqlengthsPtr, sizeof(int32_t) * seqlengths.shape(0),
                hipMemcpyHostToDevice) != hipSuccess){
        hipFree(slenCudaPtr);
        throw std::runtime_error("Cuda is out of memory");
        return 1;
    }


    //This is the Hadamard normalization constant.
    T normConstant = log2(paddedBufferSize) / 2;
    normConstant = 1 / pow(2, normConstant);

    int numRepeats = (numFreqs + paddedBufferSize - 1) / paddedBufferSize;
    int stepSize = MIN(MAX_BASE_LEVEL_TRANSFORM, paddedBufferSize);
    int log2N = log2(paddedBufferSize);

    T *featureArray;
    if (hipMalloc(&featureArray, sizeof(T) * zDim0 * paddedBufferSize) != hipSuccess) {
        hipFree(slenCudaPtr);
        hipFree(featureArray);
        throw std::runtime_error("Cuda is out of memory");
        return 1;
    };

    convRBFFeatureGenKernel<T><<<zDim0, stepSize / 2, stepSize * sizeof(T)>>>(inputPtr,
            featureArray, outputPtr, chiPtr, rademPtr, paddedBufferSize, log2N, numFreqs, zDim1, zDim2,
            numRepeats, radem.shape(2), normConstant, scalingTerm, scalingType, convWidth,
            slenCudaPtr);

    hipFree(slenCudaPtr);
    hipFree(featureArray);
    return 0;
}
//Explicitly instantiate so wrapper can use.
template int convRBFFeatureGen<double>(nb::ndarray<double, nb::shape<-1,-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<int8_t, nb::shape<3, 1, -1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<double, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        nb::ndarray<int32_t, nb::shape<-1>, nb::device::cpu, nb::c_contig> seqlengths,
        int convWidth, int scalingType);
template int convRBFFeatureGen<float>(nb::ndarray<float, nb::shape<-1,-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<int8_t, nb::shape<3, 1, -1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<float, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        nb::ndarray<int32_t, nb::shape<-1>, nb::device::cpu, nb::c_contig> seqlengths,
        int convWidth, int scalingType);



//This function generates and sums random features for an
//input array reshapedX of input type float WHILE also
//generating gradient information and storing this in
//a separate array. This gradient is only applicable
//in cases where all of the features share the same
//lengthscale; ARD-type kernels require a more complicated
//gradient calculation not implemented here.
template <typename T>
int convRBFFeatureGrad(nb::ndarray<T, nb::shape<-1,-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<int8_t, nb::shape<3, 1, -1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<T, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        nb::ndarray<int32_t, nb::shape<-1>, nb::device::cpu, nb::c_contig> seqlengths,
        nb::ndarray<double, nb::shape<-1,-1,1>, nb::device::cuda, nb::c_contig> gradArr,
        double sigma, int convWidth, int scalingType) {

    // Perform safety checks. Any exceptions thrown here are handed off to Python
    // by the Nanobind wrapper. We do not expect the user to see these because
    // the Python code will always ensure inputs are correct -- these are a failsafe
    // -- so we do not need to provide detailed exception messages here.
    int zDim0 = inputArr.shape(0);
    int zDim1 = inputArr.shape(1);
    int zDim2 = inputArr.shape(2);
    size_t numRffs = outputArr.shape(1);
    size_t numFreqs = chiArr.shape(0);
    double scalingTerm = std::sqrt(1.0 / static_cast<double>(numFreqs));

    T *inputPtr = inputArr.data();
    double *outputPtr = outputArr.data();
    T *chiPtr = chiArr.data();
    int8_t *rademPtr = radem.data();
    int32_t *seqlengthsPtr = seqlengths.data();
    double *gradientPtr = gradArr.data();

    if (inputArr.shape(0) == 0 || outputArr.shape(0) != inputArr.shape(0))
        throw std::runtime_error("no datapoints");
    if (numRffs < 2 || (numRffs & 1) != 0)
        throw std::runtime_error("last dim of output must be even number");
    if ( (2 * numFreqs) != numRffs || numFreqs > radem.shape(2) )
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    if (seqlengths.shape(0) != inputArr.shape(0))
        throw std::runtime_error("wrong array sizes");
    if (static_cast<int>(inputArr.shape(1)) < convWidth || convWidth <= 0)
        throw std::runtime_error("invalid conv_width");

    if (gradArr.shape(0) != outputArr.shape(0) || gradArr.shape(1) != outputArr.shape(1))
        throw std::runtime_error("wrong array sizes");

    double expectedNFreq = static_cast<double>(convWidth * inputArr.shape(2));
    expectedNFreq = MAX(expectedNFreq, 2);
    double log2Freqs = std::log2(expectedNFreq);
    log2Freqs = std::ceil(log2Freqs);
    int paddedBufferSize = std::pow(2, log2Freqs);

    if (radem.shape(2) % paddedBufferSize != 0)
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    int32_t minSeqLength = 2147483647, maxSeqLength = 0;
    for (size_t i=0; i < seqlengths.shape(0); i++){
        if (seqlengths(i) > maxSeqLength)
            maxSeqLength = seqlengths(i);
        if (seqlengths(i) < minSeqLength)
            minSeqLength = seqlengths(i);
    }

    if (maxSeqLength > static_cast<int32_t>(inputArr.shape(1)) || minSeqLength < convWidth){
        throw std::runtime_error("All sequence lengths must be >= conv width and < "
                "array size.");
    }

    int32_t *slenCudaPtr;
    if (hipMalloc(&slenCudaPtr, sizeof(int32_t) * seqlengths.shape(0)) != hipSuccess) {
        hipFree(slenCudaPtr);
        throw std::runtime_error("Cuda is out of memory");
        return 1;
    };
    if (hipMemcpy(slenCudaPtr, seqlengthsPtr, sizeof(int32_t) * seqlengths.shape(0),
                hipMemcpyHostToDevice) != hipSuccess){
        hipFree(slenCudaPtr);
        throw std::runtime_error("Cuda is out of memory");
        return 1;
    }




    //This is the Hadamard normalization constant.
    T normConstant = log2(paddedBufferSize) / 2;
    normConstant = 1 / pow(2, normConstant);

    int numRepeats = (numFreqs + paddedBufferSize - 1) / paddedBufferSize;
    int stepSize = MIN(MAX_BASE_LEVEL_TRANSFORM, paddedBufferSize);
    int log2N = log2(paddedBufferSize);

    T *featureArray;
    if (hipMalloc(&featureArray, sizeof(T) * zDim0 * paddedBufferSize) != hipSuccess) {
        hipFree(slenCudaPtr);
        hipFree(featureArray);
        throw std::runtime_error("Cuda is out of memory");
        return 1;
    };

    convRBFFeatureGradKernel<T><<<zDim0, stepSize / 2, stepSize * sizeof(T)>>>(inputPtr,
            featureArray, outputPtr, chiPtr, rademPtr, paddedBufferSize, log2N, numFreqs, zDim1, zDim2,
            numRepeats, radem.shape(2), normConstant, scalingTerm, scalingType, convWidth,
            slenCudaPtr, gradientPtr, sigma);

    hipFree(slenCudaPtr);
    hipFree(featureArray);
    return 0;
}
//Explicitly instantiate so wrapper can use.
template int convRBFFeatureGrad<double>(nb::ndarray<double, nb::shape<-1,-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<int8_t, nb::shape<3, 1, -1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<double, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        nb::ndarray<int32_t, nb::shape<-1>, nb::device::cpu, nb::c_contig> seqlengths,
        nb::ndarray<double, nb::shape<-1,-1,1>, nb::device::cuda, nb::c_contig> gradArr,
        double sigma, int convWidth, int scalingType);
template int convRBFFeatureGrad<float>(nb::ndarray<float, nb::shape<-1,-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<int8_t, nb::shape<3, 1, -1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<float, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        nb::ndarray<int32_t, nb::shape<-1>, nb::device::cpu, nb::c_contig> seqlengths,
        nb::ndarray<double, nb::shape<-1,-1,1>, nb::device::cuda, nb::c_contig> gradArr,
        double sigma, int convWidth, int scalingType);
