#include "hip/hip_runtime.h"
/*
* Contains specialized functions for generating random features for
* ARD RBF kernels (non-convolution).
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../shared_constants.h"
#include "../sharedmem.h"
#include "ard_ops.h"



//Performs the first piece of the gradient calculation for ARD kernels
//only -- multiplying the input data by the precomputed weight matrix
//and summing over rows that correspond to specific lengthscales.
template <typename T>
__global__ void ardGradSetup(double *gradientArray,
        T precomputedWeights[], T inputX[], int32_t *sigmaMap,
        double *sigmaVals, double *randomFeatures,
        int dim1, int numSetupElements, int numFreqs,
        int numLengthscales){

    int i, sigmaLoc;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int precompWRow = (tid % numFreqs);
    int gradRow = tid / numFreqs;

    T outVal;

    if (tid < numSetupElements){
        T *precompWElement = precomputedWeights + precompWRow * dim1;
        T *inputXElement = inputX + gradRow * dim1;
        double *gradientElement = gradientArray + 2 * (gradRow * numFreqs + precompWRow) * numLengthscales;
        double *randomFeature = randomFeatures + 2 * (gradRow * numFreqs + precompWRow);
        double rfVal = 0;

        for (i=0; i < dim1; i++){
            sigmaLoc = sigmaMap[i];
            outVal = precompWElement[i] * inputXElement[i];
            gradientElement[sigmaLoc] += outVal;
            rfVal += sigmaVals[i] * outVal;
        }
        *randomFeature = rfVal;
    }
}





//Multiplies the gradient array by the appropriate elements of the random
//feature array when calculating the gradient for ARD kernels only.
__global__ void ardGradRFMultiply(double *gradientArray, double *randomFeats,
        int numRFElements, int numFreqs, int numLengthscales,
        double rbfNormConstant){
    int i;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rowNum = tid / numFreqs, colNum = tid % numFreqs;
    int gradPosition = 2 * (rowNum * numFreqs + colNum) * numLengthscales;
    int rfPosition = 2 * (rowNum * numFreqs + colNum);
    double rfVal, cosVal, sinVal;
    

    if (tid < numRFElements){
        rfVal = randomFeats[rfPosition];
        cosVal = cos(rfVal) * rbfNormConstant;
        sinVal = sin(rfVal) * rbfNormConstant;
        randomFeats[rfPosition] = cosVal;
        randomFeats[rfPosition + 1] = sinVal;

        for (i=0; i < numLengthscales; i++){
            rfVal = gradientArray[gradPosition + i];
            gradientArray[gradPosition + i] = -rfVal * sinVal;
            gradientArray[gradPosition + i + numLengthscales] = rfVal * cosVal;
        }
    }
}


//This function generates the gradient and random features
//for ARD kernels only, using precomputed weights that take
//the place of the H-transforms
//we would otherwise need to perform.
template <typename T>
const char *ardCudaGrad(T inputX[], double *randomFeats,
                T precompWeights[], int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant){

    int numRFElements = dim0 * numFreqs;
    int numSetupElements = dim0 * numFreqs;
    int blocksPerGrid;


    blocksPerGrid = (numSetupElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    ardGradSetup<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(gradient, precompWeights, inputX,
            sigmaMap, sigmaVals, randomFeats, dim1, numSetupElements,
            numFreqs, numLengthscales);

    blocksPerGrid = (numRFElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    ardGradRFMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(gradient, randomFeats,
                numRFElements, numFreqs, numLengthscales, rbfNormConstant);

    return "no_error";
}
//Explicitly instantiate so wrappers can access.
template const char *ardCudaGrad<double>(double inputX[], double *randomFeats,
                double precompWeights[], int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant);
template const char *ardCudaGrad<float>(float inputX[], double *randomFeats,
                float precompWeights[], int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant);
