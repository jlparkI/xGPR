#include "hip/hip_runtime.h"
/*
* Contains specialized functions for generating random features for
* the RBF and related kernels (non-convolution).
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../shared_constants.h"
#include "../sharedmem.h"
#include "rbf_ops.h"


namespace nb = nanobind;


//Generates the RBF features. This single kernel loops over 1)
//the number of repeats then inside that loop 2) the three diagonal
//matrix multiplications and fast Hadamard transforms before
//applying 3) diagonal matmul before activation function.
template <typename T>
__global__ void rbfFeatureGenKernel(const T origData[], T cArray[],
        double *outputArray, const T chiArr[], const int8_t *radem,
        int paddedBufferSize, int log2N, int numFreqs, int inputElementsPerRow,
        int nRepeats, int rademShape2, T normConstant,
        double scalingConstant){
    int stepSize = MIN(paddedBufferSize, MAX_BASE_LEVEL_TRANSFORM);

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int spacing, pos = threadIdx.x;
    int lo, id1, id2;
    int tempArrPos, chiArrPos = 0;
    int inputArrPos = (blockIdx.x * inputElementsPerRow);
    int outputArrPos = (blockIdx.x * numFreqs * 2);
    T y, outputVal;
    const int8_t *rademPtr = radem;

    //Run over the number of repeats required to generate the random
    //features.
    for (int rep = 0; rep < nRepeats; rep++){
        tempArrPos = (blockIdx.x << log2N);

        //Copy original data into the temporary array.
        for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
            if (i < inputElementsPerRow)
                cArray[i + tempArrPos] = origData[i + inputArrPos];
            else
                cArray[i + tempArrPos] = 0;
        }

        //Run over three repeats for the SORF procedure.
        for (int sorfRep = 0; sorfRep < 3; sorfRep++){
            rademPtr = radem + paddedBufferSize * rep + sorfRep * rademShape2;
            tempArrPos = (blockIdx.x << log2N);

            for (int hStep = 0; hStep < paddedBufferSize; hStep+=stepSize){
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = cArray[i + tempArrPos];

                __syncthreads();

                //Multiply by the diagonal array here.
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = s_data[i] * rademPtr[i] * normConstant;

                rademPtr += stepSize;

                id1 = (pos << 1);
                id2 = id1 + 1;
                __syncthreads();
                y = s_data[id2];
                s_data[id2] = s_data[id1] - y;
                s_data[id1] += y;

                for (spacing = 2; spacing < stepSize; spacing <<= 1){
                    //Equivalent to pos mod spacing if spacing is a power of 2,
                    //which here is always true.
                    lo = pos & (spacing - 1);
                    id1 = ((pos - lo) << 1) + lo;
                    id2 = id1 + spacing;
                    __syncthreads();
                    y = s_data[id2];
                    s_data[id2] = s_data[id1] - y;
                    s_data[id1] += y;
                }
                __syncthreads();

                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    cArray[i + tempArrPos] = s_data[i];

                tempArrPos += stepSize;
                __syncthreads();
            }

            //A less efficient global memory procedure to complete the FHT
            //for long arrays.
            if (paddedBufferSize > MAX_BASE_LEVEL_TRANSFORM){
                tempArrPos = (blockIdx.x << log2N);

                for (int spacing = stepSize; spacing < paddedBufferSize; spacing <<= 1){

                    for (int k = 0; k < paddedBufferSize; k += (spacing << 1)){
                        for (int i = threadIdx.x; i < spacing; i += blockDim.x){
                            id1 = i + k + tempArrPos;
                            id2 = id1 + spacing;
                            y = cArray[id2];
                            cArray[id2] = cArray[id1] - y;
                            cArray[id1] += y;
                        }
                        __syncthreads();
                    }
                }
            }
        }
        //Now take the results stored in the temporary array, apply the
        //activation function, and populate the output array. Note that
        //we multiply by 2 in the output array position since two
        //features are generated for each frequency sampled.
        tempArrPos = (blockIdx.x << log2N);

        for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
            if ((i + chiArrPos) >= numFreqs)
                break;
            outputVal = chiArr[chiArrPos + i] * cArray[tempArrPos + i];
            outputArray[outputArrPos + 2 * i] = scalingConstant * cos(outputVal);
            outputArray[outputArrPos + 2 * i + 1] = scalingConstant * sin(outputVal);
        }

        chiArrPos += paddedBufferSize;
        outputArrPos += 2 * paddedBufferSize;
        __syncthreads();

    }
}




//Generates the RBF features with gradient. This single kernel loops over 1)
//the number of repeats then inside that loop 2) the three diagonal
//matrix multiplications and fast Hadamard transforms before
//applying 3) diagonal matmul before activation function. The only difference
//from rbfFeatureGenKernel is that the gradient is also calculated.
template <typename T>
__global__ void rbfFeatureGradKernel(const T origData[], T cArray[],
        double *outputArray, const T chiArr[], const int8_t *radem,
        int paddedBufferSize, int log2N, int numFreqs, int inputElementsPerRow,
        int nRepeats, int rademShape2, T normConstant,
        double scalingConstant, double *gradient, T sigma){
    int stepSize = MIN(paddedBufferSize, MAX_BASE_LEVEL_TRANSFORM);

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int spacing, pos = threadIdx.x;
    int lo, id1, id2;
    int tempArrPos, chiArrPos = 0;
    int inputArrPos = (blockIdx.x * inputElementsPerRow);
    int outputArrPos = (blockIdx.x * numFreqs * 2);
    T y, outputVal;
    const int8_t *rademPtr = radem;

    //Run over the number of repeats required to generate the random
    //features.
    for (int rep = 0; rep < nRepeats; rep++){
        tempArrPos = (blockIdx.x << log2N);

        //Copy original data into the temporary array.
        for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
            if (i < inputElementsPerRow)
                cArray[i + tempArrPos] = origData[i + inputArrPos];
            else
                cArray[i + tempArrPos] = 0;
        }

        //Run over three repeats for the SORF procedure.
        for (int sorfRep = 0; sorfRep < 3; sorfRep++){
            rademPtr = radem + paddedBufferSize * rep + sorfRep * rademShape2;
            tempArrPos = (blockIdx.x << log2N);

            for (int hStep = 0; hStep < paddedBufferSize; hStep+=stepSize){
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = cArray[i + tempArrPos];

                __syncthreads();

                //Multiply by the diagonal array here.
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = s_data[i] * rademPtr[i] * normConstant;

                rademPtr += stepSize;

                id1 = (pos << 1);
                id2 = id1 + 1;
                __syncthreads();
                y = s_data[id2];
                s_data[id2] = s_data[id1] - y;
                s_data[id1] += y;

                for (spacing = 2; spacing < stepSize; spacing <<= 1){
                    //Equivalent to pos mod spacing if spacing is a power of 2,
                    //which here is always true.
                    lo = pos & (spacing - 1);
                    id1 = ((pos - lo) << 1) + lo;
                    id2 = id1 + spacing;
                    __syncthreads();
                    y = s_data[id2];
                    s_data[id2] = s_data[id1] - y;
                    s_data[id1] += y;
                }
                __syncthreads();

                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    cArray[i + tempArrPos] = s_data[i];

                tempArrPos += stepSize;
                __syncthreads();
            }

            //A less efficient global memory procedure to complete the FHT
            //for long arrays.
            if (paddedBufferSize > MAX_BASE_LEVEL_TRANSFORM){
                tempArrPos = (blockIdx.x << log2N);

                for (int spacing = stepSize; spacing < paddedBufferSize; spacing <<= 1){

                    for (int k = 0; k < paddedBufferSize; k += (spacing << 1)){
                        for (int i = threadIdx.x; i < spacing; i += blockDim.x){
                            id1 = i + k + tempArrPos;
                            id2 = id1 + spacing;
                            y = cArray[id2];
                            cArray[id2] = cArray[id1] - y;
                            cArray[id1] += y;
                        }
                        __syncthreads();
                    }
                }
            }
        }
        //Now take the results stored in the temporary array, apply the
        //activation function, and populate the output array. Note that
        //we multiply by 2 in the output array position since two
        //features are generated for each frequency sampled.
        tempArrPos = (blockIdx.x << log2N);

        for (int i = threadIdx.x; i < paddedBufferSize; i += blockDim.x){
            if ((i + chiArrPos) >= numFreqs)
                break;
            outputVal = chiArr[chiArrPos + i] * cArray[tempArrPos + i];
            double prodVal = outputVal * sigma;
            outputArray[outputArrPos + 2 * i] = scalingConstant * cos(prodVal);
            outputArray[outputArrPos + 2 * i + 1] = scalingConstant * sin(prodVal);
            gradient[outputArrPos + 2 * i] = -scalingConstant * sin(prodVal) * outputVal;
            gradient[outputArrPos + 2 * i + 1] = scalingConstant * cos(prodVal) * outputVal;
        }

        chiArrPos += paddedBufferSize;
        outputArrPos += 2 * paddedBufferSize;
        __syncthreads();

    }
}




//This function generates random features for RBF / ARD kernels, if the
//input has already been multiplied by the appropriate lengthscale values.
template <typename T>
int RBFFeatureGen(
        nb::ndarray<const T, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<const int8_t, nb::shape<3,1,-1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<const T, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        bool fitIntercept) {

    // Perform safety checks. Any exceptions thrown here are handed off to Python
    // by the Nanobind wrapper. We do not expect the user to see these because
    // the Python code will always ensure inputs are correct -- these are a failsafe
    // -- so we do not need to provide detailed exception messages here.
    int zDim0 = inputArr.shape(0);
    int zDim1 = inputArr.shape(1);
    size_t numRffs = outputArr.shape(1);
    size_t numFreqs = chiArr.shape(0);
    double numFreqsFlt = numFreqs;

    const T *inputPtr = inputArr.data();
    double *outputPtr = outputArr.data();
    const T *chiPtr = chiArr.data();
    const int8_t *rademPtr = radem.data();

    if (inputArr.shape(0) == 0 || outputArr.shape(0) != inputArr.shape(0))
        throw std::runtime_error("no datapoints");
    if (numRffs < 2 || (numRffs & 1) != 0)
        throw std::runtime_error("last dim of output must be even number");
    if ( (2 * numFreqs) != numRffs || numFreqs > radem.shape(2) )
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    double expectedNFreq = (zDim1 > 2) ? static_cast<double>(zDim1) : 2.0;
    double log2Freqs = std::log2(expectedNFreq);
    log2Freqs = std::ceil(log2Freqs);
    int paddedBufferSize = std::pow(2, log2Freqs);

    if (radem.shape(2) % paddedBufferSize != 0)
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    T rbfNormConstant;

    if (fitIntercept)
        rbfNormConstant = std::sqrt(1.0 / (numFreqsFlt - 0.5));
    else
        rbfNormConstant = std::sqrt(1.0 / numFreqsFlt);

    //This is the Hadamard normalization constant.
    T normConstant = log2(paddedBufferSize) / 2;
    normConstant = 1 / pow(2, normConstant);
    int numRepeats = (numFreqs + paddedBufferSize - 1) / paddedBufferSize;
    int stepSize = MIN(MAX_BASE_LEVEL_TRANSFORM, paddedBufferSize);
    int log2N = log2(paddedBufferSize);

    T *featureArray;
    if (hipMalloc(&featureArray, sizeof(T) * zDim0 * paddedBufferSize) != hipSuccess) {
        hipFree(featureArray);
        throw std::runtime_error("out of memory on cuda");
        return 1;
    };

    rbfFeatureGenKernel<T><<<zDim0, stepSize / 2, stepSize * sizeof(T)>>>(inputPtr,
            featureArray, outputPtr, chiPtr, rademPtr, paddedBufferSize, log2N, numFreqs, zDim1,
            numRepeats, radem.shape(2), normConstant, rbfNormConstant);

    hipFree(featureArray);
    return 0;
}
//Instantiate templates so Cython / PyBind wrappers can import.
template int RBFFeatureGen<double>(
        nb::ndarray<const double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<const int8_t, nb::shape<3,1,-1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<const double, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        bool fitIntercept);
template int RBFFeatureGen<float>(
        nb::ndarray<const float, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<const int8_t, nb::shape<3,1,-1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<const float, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        bool fitIntercept);


//This function generates random features for RBF kernels ONLY
//(NOT ARD), and simultaneously generates the gradient, storing
//it in a separate array.
template <typename T>
int RBFFeatureGrad(
        nb::ndarray<const T, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<double, nb::shape<-1,-1,1>, nb::device::cuda, nb::c_contig> gradArr,
        nb::ndarray<const int8_t, nb::shape<3,1,-1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<const T, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        float sigma, bool fitIntercept) {

    // Perform safety checks. Any exceptions thrown here are handed off to Python
    // by the Nanobind wrapper. We do not expect the user to see these because
    // the Python code will always ensure inputs are correct -- these are a failsafe
    // -- so we do not need to provide detailed exception messages here.
    int zDim0 = inputArr.shape(0);
    int zDim1 = inputArr.shape(1);
    size_t numRffs = outputArr.shape(1);
    size_t numFreqs = chiArr.shape(0);
    double numFreqsFlt = numFreqs;

    const T *inputPtr = inputArr.data();
    double *outputPtr = outputArr.data();
    double *gradientPtr = gradArr.data();
    const T *chiPtr = chiArr.data();
    const int8_t *rademPtr = radem.data();

    if (inputArr.shape(0) == 0 || outputArr.shape(0) != inputArr.shape(0))
        throw std::runtime_error("no datapoints");
    if (numRffs < 2 || (numRffs & 1) != 0)
        throw std::runtime_error("last dim of output must be even number");
    if ( (2 * numFreqs) != numRffs || numFreqs > radem.shape(2) )
        throw std::runtime_error("incorrect number of rffs and or freqs.");
    if (gradArr.shape(0) != outputArr.shape(0) || gradArr.shape(1) != outputArr.shape(1))
        throw std::runtime_error("Wrong array sizes.");

    double expectedNFreq = (zDim1 > 2) ? static_cast<double>(zDim1) : 2.0;
    double log2Freqs = std::log2(expectedNFreq);
    log2Freqs = std::ceil(log2Freqs);
    int paddedBufferSize = std::pow(2, log2Freqs);

    if (radem.shape(2) % paddedBufferSize != 0)
        throw std::runtime_error("incorrect number of rffs and or freqs.");

    T rbfNormConstant;

    if (fitIntercept)
        rbfNormConstant = std::sqrt(1.0 / (numFreqsFlt - 0.5));
    else
        rbfNormConstant = std::sqrt(1.0 / numFreqsFlt);


    //This is the Hadamard normalization constant.
    T normConstant = log2(paddedBufferSize) / 2;
    normConstant = 1 / pow(2, normConstant);
    int numRepeats = (numFreqs + paddedBufferSize - 1) / paddedBufferSize;
    int stepSize = MIN(MAX_BASE_LEVEL_TRANSFORM, paddedBufferSize);
    int log2N = log2(paddedBufferSize);

    T *featureArray;
    if (hipMalloc(&featureArray, sizeof(T) * zDim0 * paddedBufferSize) != hipSuccess) {
        hipFree(featureArray);
        throw std::runtime_error("out of memory on cuda");
        return 1;
    };

    rbfFeatureGradKernel<T><<<zDim0, stepSize / 2, stepSize * sizeof(T)>>>(inputPtr,
            featureArray, outputPtr, chiPtr, rademPtr, paddedBufferSize, log2N, numFreqs, zDim1,
            numRepeats, radem.shape(2), normConstant, rbfNormConstant, gradientPtr,
            sigma);

    hipFree(featureArray);
    return 0;
}
//Instantiate templates so Cython / PyBind wrappers can import.
template int RBFFeatureGrad<double>(
        nb::ndarray<const double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<double, nb::shape<-1,-1,1>, nb::device::cuda, nb::c_contig> gradArr,
        nb::ndarray<const int8_t, nb::shape<3,1,-1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<const double, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        float sigma, bool fitIntercept);
template int RBFFeatureGrad<float>(
        nb::ndarray<const float, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> inputArr,
        nb::ndarray<double, nb::shape<-1,-1>, nb::device::cuda, nb::c_contig> outputArr,
        nb::ndarray<double, nb::shape<-1,-1,1>, nb::device::cuda, nb::c_contig> gradArr,
        nb::ndarray<const int8_t, nb::shape<3,1,-1>, nb::device::cuda, nb::c_contig> radem,
        nb::ndarray<const float, nb::shape<-1>, nb::device::cuda, nb::c_contig> chiArr,
        float sigma, bool fitIntercept);
