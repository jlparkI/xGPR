#include "hip/hip_runtime.h"
/*
* Contains specialized functions for generating random features for
* the RBF and related kernels (non-convolution).
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../shared_constants.h"
#include "../basic_ops/basic_array_operations.h"
#include "../sharedmem.h"
#include "rbf_ops.h"

//Generates the RBF features. This single kernel loops over 1)
//the number of repeats then inside that loop 2) the three diagonal
//matrix multiplications and fast Hadamard transforms before
//applying 4) the simplex projection and 5) diagonal matmul before
//activation function.
template <typename T>
__global__ void rbfFeatureGenKernel(const T origData[], T cArray[],
        double *outputArray, const T chiArr[], const int8_t *radem,
        int N, int log2N, int numFreqs, int inputElementsPerRow,
        int nRepeats, int rademShape2, T normConstant,
        double scalingConstant){
    int stepSize = MIN(N, MAX_BASE_LEVEL_TRANSFORM);

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int spacing, pos = threadIdx.x;
    int lo, id1, id2;
    int tempArrPos, chiArrPos = 0;
    int inputArrPos = (blockIdx.x * inputElementsPerRow);
    int outputArrPos = (blockIdx.x * numFreqs * 2);
    T y, outputVal;

    const int8_t *rademPtr = radem;

    //Run over the number of repeats required to generate the random
    //features.
    for (int rep = 0; rep < nRepeats; rep++){
        tempArrPos = (blockIdx.x << log2N);

        //Copy original data into the temporary array.
        for (int i = threadIdx.x; i < N; i += blockDim.x){
            if (i < inputElementsPerRow)
                cArray[i + tempArrPos] = origData[i + inputArrPos];
            else
                cArray[i + tempArrPos] = 0;
        }

        //Run over three repeats for the SORF procedure.
        for (int sorfRep = 0; sorfRep < 3; sorfRep++){
            rademPtr = radem + N * rep + sorfRep * rademShape2;
            tempArrPos = (blockIdx.x << log2N);

            for (int hStep = 0; hStep < N; hStep+=stepSize){
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = cArray[i + tempArrPos];

                __syncthreads();

                //Multiply by the diagonal array here.
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = s_data[i] * rademPtr[i] * normConstant;

                rademPtr += stepSize;

                id1 = (pos << 1);
                id2 = id1 + 1;
                __syncthreads();
                y = s_data[id2];
                s_data[id2] = s_data[id1] - y;
                s_data[id1] += y;

                for (spacing = 2; spacing < stepSize; spacing <<= 1){
                    //Equivalent to pos mod spacing if spacing is a power of 2,
                    //which here is always true.
                    lo = pos & (spacing - 1);
                    id1 = ((pos - lo) << 1) + lo;
                    id2 = id1 + spacing;
                    __syncthreads();
                    y = s_data[id2];
                    s_data[id2] = s_data[id1] - y;
                    s_data[id1] += y;
                }
                __syncthreads();

                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    cArray[i + tempArrPos] = s_data[i];

                tempArrPos += stepSize;
                __syncthreads();
            }

            //A less efficient global memory procedure to complete the FHT
            //for long arrays.
            if (N > MAX_BASE_LEVEL_TRANSFORM){
                tempArrPos = (blockIdx.x << log2N);

                for (int spacing = stepSize; spacing < N; spacing <<= 1){

                    for (int k = 0; k < N; k += (spacing << 1)){
                        for (int i = threadIdx.x; i < spacing; i += blockDim.x){
                            id1 = i + k + tempArrPos;
                            id2 = id1 + spacing;
                            y = cArray[id2];
                            cArray[id2] = cArray[id1] - y;
                            cArray[id1] += y;
                        }
                        __syncthreads();
                    }
                }
            }
        }
        //Now take the results stored in the temporary array, apply the
        //activation function, and populate the output array. Note that
        //we multiply by 2 in the output array position since two
        //features are generated for each frequency sampled.
        tempArrPos = (blockIdx.x << log2N);

        for (int i = threadIdx.x; i < N; i += blockDim.x){
            if ((i + chiArrPos) >= numFreqs)
                break;
            outputVal = chiArr[chiArrPos + i] * cArray[tempArrPos + i];
            outputArray[outputArrPos + 2 * i] = scalingConstant * cos(outputVal);
            outputArray[outputArrPos + 2 * i + 1] = scalingConstant * sin(outputVal);
        }

        chiArrPos += stepSize;
        outputArrPos += 2 * stepSize;
        __syncthreads();

    }
}



//Generates the RBF features with gradient. This single kernel loops over 1)
//the number of repeats then inside that loop 2) the three diagonal
//matrix multiplications and fast Hadamard transforms before
//applying 4) the simplex projection and 5) diagonal matmul before
//activation function. The only difference from rbfFeatureGenKernel
//is that the gradient is also calculated.
template <typename T>
__global__ void rbfFeatureGradKernel(const T origData[], T cArray[],
        double *outputArray, const T chiArr[], const int8_t *radem,
        int N, int log2N, int numFreqs, int inputElementsPerRow,
        int nRepeats, int rademShape2, T normConstant,
        double scalingConstant, double *gradient){
    int stepSize = MIN(N, MAX_BASE_LEVEL_TRANSFORM);

    SharedMemory<T> shared;
    T *s_data = shared.getPointer();
    int spacing, pos = threadIdx.x;
    int lo, id1, id2;
    int tempArrPos, chiArrPos = 0;
    int inputArrPos = (blockIdx.x * inputElementsPerRow);
    int outputArrPos = (blockIdx.x * numFreqs * 2);
    T y, outputVal;

    const int8_t *rademPtr = radem;

    //Run over the number of repeats required to generate the random
    //features.
    for (int rep = 0; rep < nRepeats; rep++){
        tempArrPos = (blockIdx.x << log2N);

        //Copy original data into the temporary array.
        for (int i = threadIdx.x; i < N; i += blockDim.x){
            if (i < inputElementsPerRow)
                cArray[i + tempArrPos] = origData[i + inputArrPos];
            else
                cArray[i + tempArrPos] = 0;
        }

        //Run over three repeats for the SORF procedure.
        for (int sorfRep = 0; sorfRep < 3; sorfRep++){
            rademPtr = radem + N * rep + sorfRep * rademShape2;
            tempArrPos = (blockIdx.x << log2N);

            for (int hStep = 0; hStep < N; hStep+=stepSize){
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = cArray[i + tempArrPos];

                __syncthreads();

                //Multiply by the diagonal array here.
                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    s_data[i] = s_data[i] * rademPtr[i] * normConstant;

                rademPtr += stepSize;

                id1 = (pos << 1);
                id2 = id1 + 1;
                __syncthreads();
                y = s_data[id2];
                s_data[id2] = s_data[id1] - y;
                s_data[id1] += y;

                for (spacing = 2; spacing < stepSize; spacing <<= 1){
                    //Equivalent to pos mod spacing if spacing is a power of 2,
                    //which here is always true.
                    lo = pos & (spacing - 1);
                    id1 = ((pos - lo) << 1) + lo;
                    id2 = id1 + spacing;
                    __syncthreads();
                    y = s_data[id2];
                    s_data[id2] = s_data[id1] - y;
                    s_data[id1] += y;
                }
                __syncthreads();

                for (int i = threadIdx.x; i < stepSize; i += blockDim.x)
                    cArray[i + tempArrPos] = s_data[i];

                tempArrPos += stepSize;
                __syncthreads();
            }

            //A less efficient global memory procedure to complete the FHT
            //for long arrays.
            if (N > MAX_BASE_LEVEL_TRANSFORM){
                tempArrPos = (blockIdx.x << log2N);

                for (int spacing = stepSize; spacing < N; spacing <<= 1){

                    for (int k = 0; k < N; k += (spacing << 1)){
                        for (int i = threadIdx.x; i < spacing; i += blockDim.x){
                            id1 = i + k + tempArrPos;
                            id2 = id1 + spacing;
                            y = cArray[id2];
                            cArray[id2] = cArray[id1] - y;
                            cArray[id1] += y;
                        }
                        __syncthreads();
                    }
                }
            }
        }
        //Now take the results stored in the temporary array, apply the
        //activation function, and populate the output array. Note that
        //we multiply by 2 in the output array position since two
        //features are generated for each frequency sampled.
        tempArrPos = (blockIdx.x << log2N);

        for (int i = threadIdx.x; i < N; i += blockDim.x){
            if ((i + chiArrPos) >= numFreqs)
                break;
            outputVal = chiArr[chiArrPos + i] * cArray[tempArrPos + i];
            outputArray[outputArrPos + 2 * i] = scalingConstant * cos(outputVal);
            outputArray[outputArrPos + 2 * i + 1] = scalingConstant * sin(outputVal);
            gradient[outputArrPos + 2 * i] = -scalingConstant * sin(outputVal) * outputVal;
            gradient[outputArrPos + 2 * i + 1] = scalingConstant * cos(outputVal) * outputVal;
        }

        chiArrPos += stepSize;
        outputArrPos += 2 * stepSize;
        __syncthreads();

    }
}



//Performs the first piece of the gradient calculation for ARD kernels
//only -- multiplying the input data by the precomputed weight matrix
//and summing over rows that correspond to specific lengthscales.
template <typename T>
__global__ void ardGradSetup(double *gradientArray,
        T precomputedWeights[], T inputX[], int32_t *sigmaMap,
        double *sigmaVals, double *randomFeatures,
        int dim1, int numSetupElements, int numFreqs,
        int numLengthscales){

    int i, sigmaLoc;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int precompWRow = (tid % numFreqs);
    int gradRow = tid / numFreqs;

    T outVal;

    if (tid < numSetupElements){
        T *precompWElement = precomputedWeights + precompWRow * dim1;
        T *inputXElement = inputX + gradRow * dim1;
        double *gradientElement = gradientArray + 2 * (gradRow * numFreqs + precompWRow) * numLengthscales;
        double *randomFeature = randomFeatures + 2 * (gradRow * numFreqs + precompWRow);
        double rfVal = 0;

        for (i=0; i < dim1; i++){
            sigmaLoc = sigmaMap[i];
            outVal = precompWElement[i] * inputXElement[i];
            gradientElement[sigmaLoc] += outVal;
            rfVal += sigmaVals[i] * outVal;
        }
        *randomFeature = rfVal;
    }
}





//Multiplies the gradient array by the appropriate elements of the random
//feature array when calculating the gradient for ARD kernels only.
__global__ void ardGradRFMultiply(double *gradientArray, double *randomFeats,
        int numRFElements, int numFreqs, int numLengthscales,
        double rbfNormConstant){
    int i;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rowNum = tid / numFreqs, colNum = tid % numFreqs;
    int gradPosition = 2 * (rowNum * numFreqs + colNum) * numLengthscales;
    int rfPosition = 2 * (rowNum * numFreqs + colNum);
    double rfVal, cosVal, sinVal;
    

    if (tid < numRFElements){
        rfVal = randomFeats[rfPosition];
        cosVal = cos(rfVal) * rbfNormConstant;
        sinVal = sin(rfVal) * rbfNormConstant;
        randomFeats[rfPosition] = cosVal;
        randomFeats[rfPosition + 1] = sinVal;

        for (i=0; i < numLengthscales; i++){
            rfVal = gradientArray[gradPosition + i];
            gradientArray[gradPosition + i] = -rfVal * sinVal;
            gradientArray[gradPosition + i + numLengthscales] = rfVal * cosVal;
        }
    }
}




//This function generates random features for RBF / ARD kernels, if the
//input has already been multiplied by the appropriate lengthscale values.
template <typename T>
const char *RBFFeatureGen(T origData[], int8_t *radem,
                T chiArr[], double *outputArray,
                double rbfNormConstant,
                int dim0, int dim1, int rademShape2,
                int numFreqs, int paddedBufferSize){
    //This is the Hadamard normalization constant.
    T normConstant = log2(paddedBufferSize) / 2;
    normConstant = 1 / pow(2, normConstant);
    int stepSize, log2N;
    int numRepeats = (numFreqs + paddedBufferSize - 1) / paddedBufferSize;

    stepSize = MIN(MAX_BASE_LEVEL_TRANSFORM, paddedBufferSize);
    log2N = log2(paddedBufferSize);

    T *featureArray;
    if (hipMalloc(&featureArray, sizeof(T) * dim0 * paddedBufferSize) != hipSuccess) {
        hipFree(featureArray);
        return "Fatal malloc error";
    };

    rbfFeatureGenKernel<T><<<dim0, stepSize / 2, stepSize * sizeof(T)>>>(origData, featureArray,
            outputArray, chiArr, radem, paddedBufferSize, log2N, numFreqs, dim1,
            numRepeats, rademShape2, normConstant, rbfNormConstant);

    hipFree(&featureArray);
    return "no_error";
}
//Instantiate templates so Cython / PyBind wrappers can import.
template const char *RBFFeatureGen<double>(double cArray[], int8_t *radem,
                double chiArr[], double *outputArray,
                double rbfNormConstant, int dim0, int dim1, 
                int rademShape2, int numFreqs, int paddedBufferSize);
template const char *RBFFeatureGen<float>(float cArray[], int8_t *radem,
                float chiArr[], double *outputArray,
                double rbfNormConstant, int dim0, int dim1,
                int rademShape2, int numFreqs, int paddedBufferSize);


//This function generates random features for RBF kernels ONLY
//(NOT ARD), and simultaneously generates the gradient, storing
//it in a separate array.
template <typename T>
const char *RBFFeatureGrad(T origData[], int8_t *radem,
                T chiArr[], double *outputArray,
                double *gradientArray, double rbfNormConstant,
                T sigma, int dim0, int dim1, int rademShape2,
                int numFreqs, int paddedBufferSize){
    //This is the Hadamard normalization constant.
    T normConstant = log2(paddedBufferSize) / 2;
    normConstant = 1 / pow(2, normConstant);
    int stepSize, log2N;
    int numRepeats = (numFreqs + paddedBufferSize - 1) / paddedBufferSize;

    stepSize = MIN(MAX_BASE_LEVEL_TRANSFORM, paddedBufferSize);
    log2N = log2(paddedBufferSize);

    T *featureArray;
    if (hipMalloc(&featureArray, sizeof(T) * dim0 * paddedBufferSize) != hipSuccess) {
        hipFree(featureArray);
        return "Fatal malloc error";
    };

    rbfFeatureGradKernel<T><<<dim0, stepSize / 2, stepSize * sizeof(T)>>>(origData, featureArray,
            outputArray, chiArr, radem, paddedBufferSize, log2N, numFreqs, dim1,
            numRepeats, rademShape2, normConstant, rbfNormConstant, gradientArray);

    hipFree(&featureArray);

    return "no_error";
}
//Instantiate templates so Cython / PyBind wrappers can import.
template const char *RBFFeatureGrad<double>(double origData[], int8_t *radem,
                double chiArr[], double *outputArray,
                double *gradientArray, double rbfNormConstant,
                double sigma, int dim0, int dim1, int rademShape2,
                int numFreqs, int paddedBufferSize);
template const char *RBFFeatureGrad<float>(float origData[], int8_t *radem,
                float chiArr[], double *outputArray,
                double *gradientArray, double rbfNormConstant,
                float sigma, int dim0, int dim1, int rademShape2,
                int numFreqs, int paddedBufferSize);


//This function generates the gradient and random features
//for ARD kernels only, using precomputed weights that take
//the place of the H-transforms
//we would otherwise need to perform.
template <typename T>
const char *ardCudaGrad(T inputX[], double *randomFeats,
                T precompWeights[], int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant){

    int numRFElements = dim0 * numFreqs;
    int numSetupElements = dim0 * numFreqs;
    int blocksPerGrid;


    blocksPerGrid = (numSetupElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    ardGradSetup<T><<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(gradient, precompWeights, inputX,
            sigmaMap, sigmaVals, randomFeats, dim1, numSetupElements,
            numFreqs, numLengthscales);

    blocksPerGrid = (numRFElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    ardGradRFMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(gradient, randomFeats,
                numRFElements, numFreqs, numLengthscales, rbfNormConstant);

    return "no_error";
}
//Explicitly instantiate so wrappers can access.
template const char *ardCudaGrad<double>(double inputX[], double *randomFeats,
                double precompWeights[], int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant);
template const char *ardCudaGrad<float>(float inputX[], double *randomFeats,
                float precompWeights[], int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant);
