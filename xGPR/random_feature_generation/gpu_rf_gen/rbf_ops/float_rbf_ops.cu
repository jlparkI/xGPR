#include "hip/hip_runtime.h"
/*
* Contains specialized functions for generating random features for
* the RBF and related kernels. It makes use of the hadamard transform functions
* implemented under array_operations.h, so only the pieces specific
* to the kernel need to be implemented here.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "../float_array_operations.h"
#include "float_rbf_ops.h"
#include <hip/hip_runtime_api.h>


#define DEFAULT_THREADS_PER_BLOCK 256
#define MAX_BASE_LEVEL_TRANSFORM 1024


//Performs the last step in the random feature generation for the
//RBF / MiniARD kernels.
__global__ void rbfFeatureGenLastStepFloats(float *cArray, double *outputArray,
            float *chiArr, int numFreqs, int inputElementsPerRow,
            int numElements, double normConstant)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int chiArrPosition, inputPosition, outputRow, outputPosition;
    float outputVal;

    chiArrPosition = j % numFreqs;
    outputRow = (j / numFreqs);
    inputPosition = outputRow * inputElementsPerRow + chiArrPosition;
    outputPosition = outputRow * 2 * numFreqs + chiArrPosition;

    outputVal = chiArr[chiArrPosition] * cArray[inputPosition];
    if (j < numElements)
    {
        outputArray[outputPosition] = normConstant * cosf(outputVal);
        outputArray[outputPosition + numFreqs] = normConstant * sinf(outputVal);
    }
}

//Performs an elementwise multiplication of a [c,M,P] array against the
//[N,M,P] input array or a [P] array against the [N,P] input array.
//Note that the last dimensions of these must be the
//same, and this function does not check this -- caller must check. Note that
//we mutiiply by the Hadamard normalization constant here.
__global__ void floatSpecMultByDiagRademMat(float *cArray, int8_t *rademArray,
			int numElementsPerRow, int numElements, float normConstant)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int rVal, position;
    
    position = j % numElementsPerRow;
    rVal = rademArray[position];
    if (j < numElements)
        cArray[j] = cArray[j] * rVal * normConstant;
}


//This function generates random features for RBF / ARD kernels, if the
//input has already been multiplied by the appropriate lengthscale values.
const char *floatRBFFeatureGen(float *cArray, int8_t *radem,
                float *chiArr, double *outputArray,
                double rbfNormConstant,
                int dim0, int dim1, int dim2,
                int numFreqs){
    int numElementsPerRow = dim1 * dim2;
    int numElements = dim1 * dim2 * dim0;
    //This is the Hadamard normalization constant.
    float normConstant = log2(dim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    int numOutputElements = numFreqs * dim0;
    //hipProfilerStart();

    //Multiply by D1.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D2.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);

    //Second H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D3.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + 2 * numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);
    
    //Last H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2); 


    //Generate output features in-place in the output array.
    blocksPerGrid = (numOutputElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    rbfFeatureGenLastStepFloats<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, outputArray,
                    chiArr, numFreqs, numElementsPerRow, numOutputElements, rbfNormConstant);

    //hipProfilerStop();
    return "no_error";
}
