#include "hip/hip_runtime.h"
/*
* Contains specialized functions for generating random features for
* the RBF and related kernels. It makes use of the hadamard transform functions
* implemented under array_operations.h, so only the pieces specific
* to the kernel need to be implemented here.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include "../basic_ops/float_array_operations.h"
#include "float_rbf_ops.h"
#include <hip/hip_runtime_api.h>


#define DEFAULT_THREADS_PER_BLOCK 256
#define MAX_BASE_LEVEL_TRANSFORM 1024



//Performs an elementwise multiplication of a [c,M,P] array against the
//[N,M,P] input array or a [P] array against the [N,P] input array.
//Note that the last dimensions of these must be the
//same, and this function does not check this -- caller must check. Note that
//we mutiiply by the Hadamard normalization constant here.
__global__ void floatSpecMultByDiagRademMat(float *cArray, int8_t *rademArray,
			int numElementsPerRow, int numElements, float normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rVal, position;
    
    position = tid % numElementsPerRow;
    rVal = rademArray[position];
    if (tid < numElements)
        cArray[tid] = cArray[tid] * rVal * normConstant;
}



//Performs the last step in the random feature generation for the
//RBF / MiniARD kernels.
__global__ void rbfFeatureGenLastStepFloats(float *cArray, double *outputArray,
            float *chiArr, int numFreqs, int inputElementsPerRow,
            int numElements, double normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int chiArrPosition, inputPosition, outputRow, outputPosition;
    float outputVal;

    chiArrPosition = tid % numFreqs;
    outputRow = (tid / numFreqs);
    inputPosition = outputRow * inputElementsPerRow + chiArrPosition;
    outputPosition = 2 * (outputRow * numFreqs + chiArrPosition);

    outputVal = chiArr[chiArrPosition] * cArray[inputPosition];
    if (tid < numElements)
    {
        outputArray[outputPosition] = normConstant * cosf(outputVal);
        outputArray[outputPosition + 1] = normConstant * sinf(outputVal);
    }
}


//Performs the last step in gradient / feature generation for RBF (NOT ARD)
//kernels.
__global__ void rbfGradLastStepFloats(float *cArray, double *outputArray,
            float *chiArr, double *gradientArray, float sigma, int numFreqs,
            int inputElementsPerRow, int numElements, double normConstant)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int chiArrPosition, inputPosition, outputRow, outputPosition;
    float outputVal, sinVal, cosVal;

    chiArrPosition = tid % numFreqs;
    outputRow = (tid / numFreqs);
    inputPosition = outputRow * inputElementsPerRow + chiArrPosition;
    outputPosition = 2 * (outputRow * numFreqs + chiArrPosition);

    outputVal = chiArr[chiArrPosition] * cArray[inputPosition];
    if (tid < numElements)
    {
        cosVal = normConstant * cosf(outputVal * sigma);
        sinVal = normConstant * sinf(outputVal * sigma);
        outputArray[outputPosition] = cosVal;
        outputArray[outputPosition + 1] = sinVal;
        gradientArray[outputPosition] = -outputVal * sinVal;
        gradientArray[outputPosition + 1] = outputVal * cosVal;
    }
}



//Performs the first piece of the gradient calculation for ARD kernels
//only -- multiplying the input data by the precomputed weight matrix
//and summing over rows that correspond to specific lengthscales.
__global__ void ardFloatGradSetup(double *gradientArray,
        float *precomputedWeights, float *inputX, int32_t *sigmaMap,
        double *sigmaVals, double *randomFeatures,
        int dim1, int numSetupElements,
        int numFreqs, int numLengthscales){

    int i, sigmaLoc;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int precompWRow = (tid % numFreqs);
    int gradRow = tid / numFreqs;

    float *precompWElement = precomputedWeights + precompWRow * dim1;
    float *inputXElement = inputX + gradRow * dim1;
    double *gradientElement = gradientArray + 2 * (gradRow * numFreqs + precompWRow) * numLengthscales;
    double *randomFeature = randomFeatures + 2 * (gradRow * numFreqs + precompWRow);
    double rfVal = 0;
    float outVal;

    if (tid < numSetupElements){
        for (i=0; i < dim1; i++){
            sigmaLoc = sigmaMap[i];
            outVal = precompWElement[i] * inputXElement[i];
            gradientElement[sigmaLoc] += outVal;
            rfVal += sigmaVals[i] * outVal;
        }
        *randomFeature = rfVal;
    }
}





//Multiplies the gradient array by the appropriate elements of the random
//feature array when calculating the gradient for ARD kernels only.
__global__ void ardFloatGradRFMultiply(double *gradientArray, double *randomFeats,
        int numRFElements, int numFreqs, int numLengthscales,
        double rbfNormConstant){
    int i;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rowNum = tid / numFreqs, colNum = tid % numFreqs;
    int gradPosition = 2 * (rowNum * numFreqs + colNum) * numLengthscales;
    int rfPosition = 2 * (rowNum * numFreqs + colNum);
    double rfVal, cosVal, sinVal;
    

    if (tid < numRFElements){
        rfVal = randomFeats[rfPosition];
        cosVal = cos(rfVal) * rbfNormConstant;
        sinVal = sin(rfVal) * rbfNormConstant;
        randomFeats[rfPosition] = cosVal;
        randomFeats[rfPosition + 1] = sinVal;

        for (i=0; i < numLengthscales; i++){
            rfVal = gradientArray[gradPosition + i];
            gradientArray[gradPosition + i] = -rfVal * sinVal;
            gradientArray[gradPosition + i + numLengthscales] = rfVal * cosVal;
        }
    }
}




//This function generates random features for RBF / ARD kernels, if the
//input has already been multiplied by the appropriate lengthscale values.
const char *floatRBFFeatureGen(float *cArray, int8_t *radem,
                float *chiArr, double *outputArray,
                double rbfNormConstant,
                int dim0, int dim1, int dim2,
                int numFreqs){
    int numElementsPerRow = dim1 * dim2;
    int numElements = dim1 * dim2 * dim0;
    //This is the Hadamard normalization constant.
    float normConstant = log2(dim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    int numOutputElements = numFreqs * dim0;
    //hipProfilerStart();

    //Multiply by D1.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D2.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);

    //Second H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D3.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + 2 * numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);
    
    //Last H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2); 


    //Generate output features in-place in the output array.
    blocksPerGrid = (numOutputElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    rbfFeatureGenLastStepFloats<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, outputArray,
                    chiArr, numFreqs, numElementsPerRow, numOutputElements, rbfNormConstant);

    //hipProfilerStop();
    return "no_error";
}



//This function generates random features for RBF kernels ONLY
//(NOT ARD), and simultaneously generates the gradient, storing
//it in a separate array.
const char *floatRBFFeatureGrad(float *cArray, int8_t *radem,
                float *chiArr, double *outputArray,
                double *gradientArray, double rbfNormConstant,
                float sigma, int dim0, int dim1, int dim2,
                int numFreqs){
    int numElementsPerRow = dim1 * dim2;
    int numElements = dim1 * dim2 * dim0;
    //This is the Hadamard normalization constant.
    float normConstant = log2(dim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    int numOutputElements = numFreqs * dim0;
    //hipProfilerStart();

    //Multiply by D1.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D2.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);

    //Second H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D3.
    floatSpecMultByDiagRademMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + 2 * numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);
    
    //Last H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2); 


    //Generate output features in-place in the output array.
    blocksPerGrid = (numOutputElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    rbfGradLastStepFloats<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, outputArray,
                    chiArr, gradientArray, sigma, numFreqs,
                    numElementsPerRow, numOutputElements, rbfNormConstant);

    //hipProfilerStop();
    return "no_error";
}



//This function generates the gradient and random features
//for ARD kernels only, using precomputed weights that take
//the place of the H-transforms
//we would otherwise need to perform.
const char *ardCudaFloatGrad(float *inputX, double *randomFeats,
                float *precompWeights, int32_t *sigmaMap,
                double *sigmaVals, double *gradient, int dim0,
                int dim1, int numLengthscales, int numFreqs,
                double rbfNormConstant){

    int numRFElements = dim0 * numFreqs;
    int numSetupElements = dim0 * numFreqs;
    int blocksPerGrid;


    blocksPerGrid = (numSetupElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    ardFloatGradSetup<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(gradient, precompWeights, inputX,
            sigmaMap, sigmaVals, randomFeats, dim1, numSetupElements,
            numFreqs, numLengthscales);

    blocksPerGrid = (numRFElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    ardFloatGradRFMultiply<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(gradient, randomFeats,
                numRFElements, numFreqs, numLengthscales, rbfNormConstant);

    return "no_error";
}
