#include "hip/hip_runtime.h"
/*
* Contains all functions needed to run the structured orthogonal features
* (SORF) operation on an input 3d array of floats on GPU. The input array should
* already live on GPU.
*/

//Note that where possible, we have
//avoided use of modulo and integer division
//because they are more expensive. Instead, we use
//(location >> log2Spacing) (equivalent to floor division on
//location / spacing IF spacing is a power of 2) and 
//(location & (spacing - 1)) 
//(equivalent to location % spacing IF spacing is a power of 2). 
//Note that all of this like many operations here works ONLY
//if array.shape[2] is a power of 2 -- this should
//ALWAYS be checked by caller. The Cython wrapper checks this,
//if you decide to use this outside of the Cython wrapper,
//you must check yourself.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "float_array_operations.h"
#include <hip/hip_runtime_api.h>


#define DEFAULT_THREADS_PER_BLOCK 256
#define MAX_BASE_LEVEL_TRANSFORM 1024



//Uses shared memory to perform a reasonably efficient single kernel
//transform covering strides up to MAX_BASE_LEVEL_TRANSFORM.
__global__ void floatBaseLevelTransform(float *cArray, int N, int log2N){
    int startPos = blockIdx.x << log2N;

    extern __shared__ float s_data[];
    int i, spacing, pos = threadIdx.x;
    int lo, id1, id2;
    float *src_ptr = cArray + startPos;
    float y;

    for (i = threadIdx.x; i < N; i += blockDim.x)
        s_data[i] = src_ptr[i];

    id1 = (pos << 1);
    id2 = id1 + 1;
    __syncthreads();
    y = s_data[id2];
    s_data[id2] = s_data[id1] - y;
    s_data[id1] += y;


    for (spacing = 2; spacing < N; spacing <<= 1){
        lo = pos & (spacing - 1);
        id1 = ((pos - lo) << 1) + lo;
        id2 = id1 + spacing;
        __syncthreads();
        y = s_data[id2];
        s_data[id2] = s_data[id1] - y;
        s_data[id1] += y;
    }
    for (i = threadIdx.x; i < N; i += blockDim.x){
        src_ptr[i] = s_data[i];
    }
}

//Combines the level 2 transform and the level 4 transform in
//global memory for arrays with small sizes where shape[2] < 32
//so that the shared memory procedure is not efficient.
__global__ void floatShape4Transform(float *cArray, int arrsize)
{
    int id = 4 * (blockDim.x * blockIdx.x + threadIdx.x);
    int id1 = id, id2 = id + 1;
    float y;
    if (id < arrsize){
        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
        id1 += 2;
        id2 += 2;

        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
        id1 -= 1;

        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
        id1 -= 1;
        id2 -= 1;

        y = cArray[id2];
        cArray[id2] = cArray[id1] - y;
        cArray[id1] += y;
    }
}



//Performs subsequent stages of the transform (strides
//> MAX_BASE_LEVEL_TRANSFORM) using a less efficient global
//memory procedure.
__global__ void floatLevelNTransform(float *cArray, int arrsize,
                                int spacing)
{
    int pos = blockDim.x * blockIdx.x + threadIdx.x;
    int lo = (pos & (spacing - 1));
    int id = lo + ((pos - lo) << 1);
    float y, *cPtr = cArray + id;
    
    if (id < arrsize){
        y = cPtr[spacing];
        cPtr[spacing] = *cPtr - y;
        *cPtr += y;
    }
}


//Performs an elementwise multiplication of a [c,M,P] array against the
//[N,M,P] input array or a [P] array against the [N,P] input array.
//Note that the last dimensions of these must be the
//same, and this function does not check this -- caller must check. Note that
//we mutiiply by the Hadamard normalization constant here.
__global__ void floatMultiplyByDiagonalRademacherMat(float *cArray, int8_t *rademArray,
			int numElementsPerRow, int numElements, float normConstant)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int rVal, position;
    
    position = j % numElementsPerRow;
    rVal = rademArray[position];
    if (j < numElements)
        cArray[j] = cArray[j] * rVal * normConstant;
}


//Performs an elementwise multiplication of a [c,M,P] array against the
//[N,M,P] input array or a [P] array against the [N,P] input array.
//Note that the last dimensions of these must be the
//same, and this function does not check this -- caller must check. Note that
//we mutiiply by the Hadamard normalization constant here.
__global__ void floatMultiplyByDiagonalMat(float *cArray, float *diagArray,
			int numElementsPerRow, int numElements, float normConstant)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    float rVal;
    int position;
    
    position = j % numElementsPerRow;
    rVal = diagArray[position];
    if (j < numElements)
        cArray[j] *= rVal * normConstant;
}



//This function performs the SORF block transform (HD3 HD2 HD1) 
//Note that cArray must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria.
const char *floatCudaSORF3d(float *cArray, int8_t *radem,
                int dim0, int dim1, int dim2){
    int numElementsPerRow = dim1 * dim2;
    int numElements = dim1 * dim2 * dim0;
    //This is the Hadamard normalization constant.
    float normConstant = log2(dim2) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    //hipProfilerStart();

    //Multiply by D1.
    floatMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D2.
    floatMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);

    //Second H-transform.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2);
    
    //Multiply by D3.
    floatMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem + 2 * numElementsPerRow,
                                 numElementsPerRow, numElements, normConstant);
    
    //Last H-transform. Transform is in place so do not need to return anything except no error message.
    floatCudaHTransform3d(cArray, dim0, dim1, dim2); 

    //hipProfilerStop();
    return "no_error";
}




//Performs the first two steps of SRHT (HD)
//Note that cArray must have the same size across the
//last two dimensions as radem and its last dimension must
//be a power of two -- if those conditions are not met, you may
//get an unpredictable result! The Cython wrapper checks all
//of these criteria -- any other caller using this function
//should do the same.
const char *floatCudaSRHT2d(float *cArray, int8_t *radem,
                int dim0, int dim1){
    int numElementsPerRow = dim1;
    int numElements = dim1 * dim0;
    //This is the Hadamard normalization constant.
    float normConstant = log2(dim1) / 2;
    normConstant = 1 / pow(2, normConstant);
    int blocksPerGrid = (numElements + DEFAULT_THREADS_PER_BLOCK - 1) / DEFAULT_THREADS_PER_BLOCK;
    //hipProfilerStart();

    //Multiply by D1.
    floatMultiplyByDiagonalRademacherMat<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, radem, 
                                 numElementsPerRow, numElements, normConstant);
    
    //First H-transform.
    floatCudaHTransform2d(cArray, dim0, dim1);

    //hipProfilerStop();
    return "no_error";
}




//We perform the transform over the last dimension
//of cArray which must be 3d; we expect cArray.shape[2] to 
//be a power of 2 (caller must verify).
void floatCudaHTransform3d(float *cArray,
		int dim0, int dim1, int dim2){

    int N, log2N;
    int spacing = 1;
    int arrsize = dim0 * dim1 * dim2;
    int blocksPerGrid;

    //For less than 32, use specialized routines. dim2 is always
    //a power of two, and for best performance on CUDA threads per block
    //should be a multiple of 32, so the baseLevelTransform does
    //not work as well for dim2 < 32. There is a great deal of room
    //for additional optimization here that we have not done (yet) 
    //because input dim < 32 but > 2 is a somewhat niche application.
    if (dim2 < 32){
        blocksPerGrid = getNumBlocksFloatTransform(arrsize, 2);
        if (dim2 == 2){
            floatLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, 1);
        }
        else{
            floatShape4Transform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize);
            spacing = 4;
            while (spacing < dim2){
                floatLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
                spacing <<= 1;
            }
        }
        return;
    }

    //Otherwise, we use the baseLevelTransform, which uses shared
    //memory and is relatively efficient. baseLevelTransform only
    //covers strides up to MAX_BASE_LEVEL_TRANSFORM. If dim2 is less than that,
    //we're set. Otherwise, run baseLevelTransform first then use
    //a somewhat slower global memory procedure for larger strides.
    N = (MAX_BASE_LEVEL_TRANSFORM < dim2) ? MAX_BASE_LEVEL_TRANSFORM : dim2;
    log2N = log2(N);
    blocksPerGrid = arrsize / N;

    floatBaseLevelTransform<<<blocksPerGrid, N / 2, 
                    N * sizeof(float)>>>(cArray, N, log2N);
    
    if (dim2 <= MAX_BASE_LEVEL_TRANSFORM)
        return;
    
    //The largest strides (for large dim2) are handled by a somewhat
    //slower global memory procedure.
    spacing = MAX_BASE_LEVEL_TRANSFORM;
    blocksPerGrid = getNumBlocksFloatTransform(arrsize, 2);
    while (spacing < dim2){
        floatLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
        spacing <<= 1;
    }
}




//We perform the transform over the last dimension
//of cArray which must be 2d; we expect cArray.shape[1] to 
//be a power of 2 (caller must verify).
void floatCudaHTransform2d(float *cArray,
		int dim0, int dim1){

    int N, log2N;
    int spacing = 1;
    int arrsize = dim0 * dim1;
    int blocksPerGrid;

    //For less than 64, use specialized routines. dim2 is always
    //a power of two, and for best performance on CUDA threads per block
    //should be a multiple of 32, so the baseLevelTransform does
    //not work as well for dim2 < 64. There is a great deal of room
    //for additional optimization here that we have not done (yet) 
    //because input dim < 32 but > 2 is a somewhat niche application.
    if (dim1 < 32){
        blocksPerGrid = getNumBlocksFloatTransform(arrsize, 2);
        if (dim1 == 2){
            floatLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, 1);
        }
        else{
            floatShape4Transform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize);
            spacing = 4;
            while (spacing < dim1){
                floatLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
                spacing <<= 1;
            }
        }
        return;
    }

    //Otherwise, we use the baseLevelTransform, which uses shared
    //memory and is relatively efficient. baseLevelTransform only
    //covers strides up to MAX_BASE_LEVEL_TRANSFORM. If dim2 is less than that,
    //we're set. Otherwise, run baseLevelTransform first then use
    //a somewhat slower global memory procedure for larger strides.
    N = (MAX_BASE_LEVEL_TRANSFORM < dim1) ? MAX_BASE_LEVEL_TRANSFORM : dim1;
    log2N = log2(N);
    blocksPerGrid = arrsize / N;

    floatBaseLevelTransform<<<blocksPerGrid, N / 2, 
                    N * sizeof(float)>>>(cArray, N, log2N);
    
    if (dim1 <= MAX_BASE_LEVEL_TRANSFORM)
        return;
    
    //The largest strides (for large dim1) are handled by a somewhat
    //slower global memory procedure.
    spacing = MAX_BASE_LEVEL_TRANSFORM;
    blocksPerGrid = getNumBlocksFloatTransform(arrsize, 2);
    while (spacing < dim1){
        floatLevelNTransform<<<blocksPerGrid, DEFAULT_THREADS_PER_BLOCK>>>(cArray, 
                                arrsize, spacing);
        spacing <<= 1;
    }
}




//Calculates the number of blocks for all transforms except the 
//baseLevelTransform, which uses shared memory and hence
//a slightly different procedure.
int getNumBlocksFloatTransform(int arrsize, int divisor){

    int blocksPerGrid;
    blocksPerGrid = (arrsize / divisor) + DEFAULT_THREADS_PER_BLOCK - 1;
    return blocksPerGrid / DEFAULT_THREADS_PER_BLOCK;
}
